#include "hip/hip_runtime.h"
#include "MWDepthConcatenationLayerImpl.hpp"
#include "MWDepthConcatenationLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include <stdarg.h>
#include <cassert>
 MWDepthConcatenationLayerImpl::MWDepthConcatenationLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl, int outbufIdx) : MWCNNLayerImpl(layer, 
ntwk_impl) { createDepthConcatenationLayer(outbufIdx); } 
MWDepthConcatenationLayerImpl::~MWDepthConcatenationLayerImpl() {  } void 
MWDepthConcatenationLayerImpl::createDepthConcatenationLayer(int outbufIdx) {  
MWTensor* opTensor = getLayer()->getOutputTensor(0);  if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*opTensor->getHeight()*opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); 
} else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(0))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
__global__ __launch_bounds__(1024) concatImpl(float* in, float* out, size_t 
numElems,  size_t batchSize,  size_t outStride,  size_t startOffset)  { size_t 
i = blockDim.x * blockIdx.x + threadIdx.x; size_t maxElems = 
numElems*batchSize; for (; i < maxElems; i += size_t(blockDim.x*gridDim.x)) { 
size_t batchOffset = i/numElems; size_t elemOffset = i - 
(batchOffset*numElems);  int outOffset = startOffset + batchOffset*outStride; 
out[elemOffset + outOffset] = in[i];  } } void 
MWDepthConcatenationLayerImpl::predict() { int outputOffset = 0; MWTensor* 
opTensor = getLayer()->getOutputTensor(0); int outputStridePerBatch = 
opTensor->getHeight()*opTensor->getWidth()*opTensor->getChannels(); for (int k 
= 0; k < getLayer()->getNumInputs(); k++) { MWTensor* ipTensor = 
getLayer()->getInputTensor(k); int fSKMHAqIghbYYgyIpNDw = 
ipTensor->getBatchSize()* ipTensor->getHeight()* ipTensor->getWidth()* 
ipTensor->getChannels();  int tnTPxeDjBsqLAPkJcPJX = 
std::ceil(static_cast<float>(fSKMHAqIghbYYgyIpNDw)/static_cast<float>(32))*32; 
tnTPxeDjBsqLAPkJcPJX = (tnTPxeDjBsqLAPkJcPJX < 1024) ? 
tnTPxeDjBsqLAPkJcPJX : 1024; int MNuwXDSoGEYeABeVTwOh = 
(fSKMHAqIghbYYgyIpNDw + tnTPxeDjBsqLAPkJcPJX - 
1)/tnTPxeDjBsqLAPkJcPJX; int numElemsPerBatch = 
ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels(); 
concatImpl<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>(ipTensor->getData(), getData(), numElemsPerBatch, 
ipTensor->getBatchSize(), outputStridePerBatch, outputOffset); outputOffset += 
numElemsPerBatch; } } void MWDepthConcatenationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(0))); 
if(PtkeOkuClHzhOfpmBevf) { if(getLayer()->getOutputTensor(0)->getopBufIndex() < 0) 
CUDA_FREE_CALL(PtkeOkuClHzhOfpmBevf); } }