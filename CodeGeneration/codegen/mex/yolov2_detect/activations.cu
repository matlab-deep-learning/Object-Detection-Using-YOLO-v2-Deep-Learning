#include "hip/hip_runtime.h"
/*
 * activations.cu
 *
 * Code generation for function 'activations'
 *
 */

/* Include files */
#include "MWCudaDimUtility.h"
#include "rt_nonfinite.h"
#include "yolov2_detect.h"
#include "activations.h"
#include "DeepLearningNetwork.h"

/* Function Declarations */
static __global__ void b_DeepLearningNetwork_activatio(const real32_T inputdata
  [150528], real32_T inputT[150528]);
static __global__ void c_DeepLearningNetwork_activatio(real32_T out[4704],
  real32_T outT[4704]);

/* Function Definitions */
static __global__ __launch_bounds__(512, 1) void b_DeepLearningNetwork_activatio
  (const real32_T inputdata[150528], real32_T inputT[150528])
{
  uint32_T threadId;
  int32_T i4;
  int32_T i5;
  int32_T p;
  uint32_T tmpIndex;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i4 = static_cast<int32_T>((threadId % 224U));
  tmpIndex = (threadId - static_cast<uint32_T>(i4)) / 224U;
  i5 = static_cast<int32_T>((tmpIndex % 224U));
  tmpIndex = (tmpIndex - static_cast<uint32_T>(i5)) / 224U;
  p = static_cast<int32_T>(tmpIndex);
  if (p < 3) {
    inputT[(i4 + 224 * i5) + 50176 * p] = inputdata[(i5 + 224 * i4) + 50176 * p];
  }
}

static __global__ __launch_bounds__(512, 1) void c_DeepLearningNetwork_activatio
  (real32_T out[4704], real32_T outT[4704])
{
  uint32_T threadId;
  int32_T i4;
  int32_T i5;
  int32_T p;
  uint32_T tmpIndex;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i4 = static_cast<int32_T>((threadId % 14U));
  tmpIndex = (threadId - static_cast<uint32_T>(i4)) / 14U;
  i5 = static_cast<int32_T>((tmpIndex % 14U));
  tmpIndex = (tmpIndex - static_cast<uint32_T>(i5)) / 14U;
  p = static_cast<int32_T>(tmpIndex);
  if (p < 24) {
    outT[(i4 + 14 * i5) + 196 * p] = out[(i5 + 14 * i4) + 196 * p];
  }
}

void DeepLearningNetwork_activations(b_Yolov2UsingResNet50_ONNX_0 *obj, const
  real32_T inputdata[150528], real32_T outT[4704])
{
  real32_T (*gpu_inputT)[150528];
  real32_T (*gpu_out)[4704];
  real32_T (*gpu_inputdata)[150528];
  real32_T (*gpu_outT)[4704];
  hipMalloc(&gpu_outT, 18816ULL);
  hipMalloc(&gpu_out, 18816ULL);
  hipMalloc(&gpu_inputT, 602112ULL);
  hipMalloc(&gpu_inputdata, 602112ULL);
  hipMemcpy(gpu_inputdata, (void *)&inputdata[0], 602112ULL,
             hipMemcpyHostToDevice);
  b_DeepLearningNetwork_activatio<<<dim3(294U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_inputdata, *gpu_inputT);
  hipMemcpy(obj->inputData, *gpu_inputT, 150528U * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  obj->activations(153);
  hipMemcpy(*gpu_out, obj->getLayerOutput(153, 0), 4704U * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  c_DeepLearningNetwork_activatio<<<dim3(10U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_out, *gpu_outT);
  hipMemcpy(&outT[0], gpu_outT, 18816ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_inputdata);
  hipFree(*gpu_inputT);
  hipFree(*gpu_out);
  hipFree(*gpu_outT);
}

/* End of code generation (activations.cu) */
