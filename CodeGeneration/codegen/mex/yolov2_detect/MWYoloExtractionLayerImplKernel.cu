#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
 void __global__ __launch_bounds__(1024) YoloExtractionImpl(float* inputBuffer, 
float* outputBuffer_iouxy, float* outputBuffer_wh, float* outputBuffer_cscores, 
int eybNKlJCSDUvsznWynwK, int aFDPITUhkPdupMfPOBnd, long int 
ZinudJuZuGitiNTsJpBR, long int ZUTPCvgISoRdtnhGqXzM, long int 
jhFUWlztBndwjbXwYNaJ, long int jmcFOAbZArjGDNhshSro, 
long int jaqKGCwoANNDMHgAsehk, const long int BUOdotSvmFyUWQKMUdra) { 
for (int idx = blockDim.x * blockIdx.x + threadIdx.x; idx < BUOdotSvmFyUWQKMUdra; idx 
+= blockDim.x * gridDim.x) { int eVAFqeShtGZAZluKdMvQ = idx / 
ZUTPCvgISoRdtnhGqXzM; long int FLuSVNoPhAFKtLUchSvv = idx - 
(ZUTPCvgISoRdtnhGqXzM * eVAFqeShtGZAZluKdMvQ); int NNhshzQGJHLSGjDiVerE = 
static_cast<int>(FLuSVNoPhAFKtLUchSvv / ZinudJuZuGitiNTsJpBR); long 
int FeVcBgtQmTLtmnNcJGMY = FLuSVNoPhAFKtLUchSvv - 
(ZinudJuZuGitiNTsJpBR * NNhshzQGJHLSGjDiVerE); int yCdIUfwoZFngCRRRkCTg = 
static_cast<int>(FeVcBgtQmTLtmnNcJGMY % aFDPITUhkPdupMfPOBnd); int 
wvufwFZlsnpjbxmTBVYE = static_cast<int>(FeVcBgtQmTLtmnNcJGMY / aFDPITUhkPdupMfPOBnd); 
if (NNhshzQGJHLSGjDiVerE < 3 * eybNKlJCSDUvsznWynwK) { long int opIdx = eVAFqeShtGZAZluKdMvQ * 
jhFUWlztBndwjbXwYNaJ + NNhshzQGJHLSGjDiVerE * 
ZinudJuZuGitiNTsJpBR + wvufwFZlsnpjbxmTBVYE * aFDPITUhkPdupMfPOBnd + 
yCdIUfwoZFngCRRRkCTg; outputBuffer_iouxy[opIdx] = inputBuffer[idx]; } else if 
(NNhshzQGJHLSGjDiVerE >= 3 * eybNKlJCSDUvsznWynwK && NNhshzQGJHLSGjDiVerE < 5 * 
eybNKlJCSDUvsznWynwK) { int NZjOkZPwLzQsdEVkwMcX = NNhshzQGJHLSGjDiVerE - (3 * 
eybNKlJCSDUvsznWynwK); long int opIdx = eVAFqeShtGZAZluKdMvQ * 
jmcFOAbZArjGDNhshSro + NZjOkZPwLzQsdEVkwMcX * 
ZinudJuZuGitiNTsJpBR + wvufwFZlsnpjbxmTBVYE * aFDPITUhkPdupMfPOBnd + 
yCdIUfwoZFngCRRRkCTg; outputBuffer_wh[opIdx] = inputBuffer[idx]; } else { int 
NZjOkZPwLzQsdEVkwMcX = NNhshzQGJHLSGjDiVerE - (5 * eybNKlJCSDUvsznWynwK); long int opIdx = 
eVAFqeShtGZAZluKdMvQ * jaqKGCwoANNDMHgAsehk + NZjOkZPwLzQsdEVkwMcX * 
ZinudJuZuGitiNTsJpBR + wvufwFZlsnpjbxmTBVYE * aFDPITUhkPdupMfPOBnd + 
yCdIUfwoZFngCRRRkCTg; outputBuffer_cscores[opIdx] = inputBuffer[idx]; } } }