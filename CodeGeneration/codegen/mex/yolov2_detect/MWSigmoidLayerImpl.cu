#include "MWSigmoidLayer.hpp"
#include "MWSigmoidLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include <stdarg.h>
#include <cassert>
 MWSigmoidLayerImpl::MWSigmoidLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int outbufIdx) : MWCNNLayerImpl(layer, ntwk_impl)  { 
createSigmoidLayer(outbufIdx); } MWSigmoidLayerImpl::~MWSigmoidLayerImpl() { } 
void MWSigmoidLayerImpl::createSigmoidLayer(int outbufIdx) { MWSigmoidLayer* 
SigmoidLayer = static_cast<MWSigmoidLayer*>(getLayer()); MWTensor* ipTensor = 
SigmoidLayer->getInputTensor(0); MWTensor* opTensor = 
SigmoidLayer->getOutputTensor(0); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*opTensor->getHeight()*opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); 
} else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(0))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW,  HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(),  
ipTensor->getChannels(),  ipTensor->getHeight(),  ipTensor->getWidth()));  
CUDNN_CALL(hipdnnCreateActivationDescriptor(&pckLLTEdVPoCZLRwyDnM)); 
CUDNN_CALL(hipdnnSetActivationDescriptor(pckLLTEdVPoCZLRwyDnM, 
HIPDNN_ACTIVATION_SIGMOID,  HIPDNN_NOT_PROPAGATE_NAN, 0));  } void 
MWSigmoidLayerImpl::predict() { MWSigmoidLayer* SigmoidLayer = 
static_cast<MWSigmoidLayer*>(getLayer()); MWTensor* ipTensor = 
SigmoidLayer->getInputTensor(0);  MWTensor* opTensor = 
SigmoidLayer->getOutputTensor(0); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnActivationForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
pckLLTEdVPoCZLRwyDnM,  getOnePtr(), ipDesc, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(), opTensor->getData())); } void 
MWSigmoidLayerImpl::cleanup() { MWSigmoidLayer* SigmoidLayer = 
static_cast<MWSigmoidLayer*>(getLayer()); 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(pckLLTEdVPoCZLRwyDnM)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(0))); MWTensor* op 
= SigmoidLayer->getOutputTensor(); float* data = op->getData(); if (data) { 
if(getLayer()->getOutputTensor()->getopBufIndex() < 0) CUDA_FREE_CALL(data); } }