#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
 void __global__ __launch_bounds__(1024) exp_kernel(float* inputBuffer, float* 
outputBuffer, const long int BUOdotSvmFyUWQKMUdra) { for (int idx = blockDim.x * 
blockIdx.x + threadIdx.x; idx < BUOdotSvmFyUWQKMUdra; idx += blockDim.x * gridDim.x) { 
outputBuffer[idx] = exp(inputBuffer[idx]); } }