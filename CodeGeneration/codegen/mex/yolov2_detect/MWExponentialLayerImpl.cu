#include "MWExponentialLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <cassert>
 MWExponentialLayerImpl::MWExponentialLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int outbufIdx) : MWCNNLayerImpl(layer, 
ntwk_impl) { createExponentialLayer(outbufIdx); } 
MWExponentialLayerImpl::~MWExponentialLayerImpl() { } void 
MWExponentialLayerImpl::createExponentialLayer(int outbufIdx) { MWTensor* 
opTensor = getLayer()->getOutputTensor(0); if (outbufIdx < 0) { float* buffer = 
NULL; CUDA_CALL(hipMalloc((void**)&buffer, sizeof(float) * 
opTensor->getBatchSize() * opTensor->getChannels() * opTensor->getHeight() * 
opTensor->getWidth())); opTensor->setData(buffer); } else { 
opTensor->setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor( *getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } void 
MWExponentialLayerImpl::predict() { assert(this->getData() != 
getLayer()->getInputTensor(0)->getData()); long int iPqBiFnIJMxelVhQBZex = 
getLayer()->getOutputTensor(0)->getHeight() * 
getLayer()->getOutputTensor(0)->getWidth() * 
getLayer()->getOutputTensor(0)->getChannels() * 
getLayer()->getOutputTensor(0)->getBatchSize(); long int rSmEWccbJFfPGddhPemm = 
((iPqBiFnIJMxelVhQBZex + 31) / 32) * 32; long int tnTPxeDjBsqLAPkJcPJX = 
(rSmEWccbJFfPGddhPemm < 1024) ? rSmEWccbJFfPGddhPemm : 1024; long int 
MNuwXDSoGEYeABeVTwOh = (iPqBiFnIJMxelVhQBZex + tnTPxeDjBsqLAPkJcPJX - 
1) / tnTPxeDjBsqLAPkJcPJX; exp_kernel<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>( getLayer()->getInputTensor(0)->getData(), 
getLayer()->getOutputTensor(0)->getData(), iPqBiFnIJMxelVhQBZex); return; } 
void MWExponentialLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for (int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) { float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { if 
(getLayer()->getOutputTensor(idx)->getopBufIndex() == -1) { 
CUDA_FREE_CALL(data); } } } }