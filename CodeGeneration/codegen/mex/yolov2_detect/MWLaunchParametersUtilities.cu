#include "hip/hip_runtime.h"
/* Copyright 2017-2018 The MathWorks, Inc. */
#include "MWLaunchParametersUtilities.h"

bool mwGetLaunchParameters(double numberOfThreads,
                           dim3 *grid, dim3 *block,
                           MW_LAUNCH_UNSIGNED_TYPE MAX_THREADS_PER_BLOCK,
                           MW_LAUNCH_UNSIGNED_TYPE MAX_BLOCKS_PER_GRID_DIM) {

    if ( numberOfThreads <= 0.0 )
    {
        mwResetDim3ToZeros(grid);
        mwResetDim3ToZeros(block);
        return false;
    }

    if ( numberOfThreads < (double)( MAX_THREADS_PER_BLOCK * MAX_BLOCKS_PER_GRID_DIM) ) {

        return mwGetLaunchParameters1D(numberOfThreads,
                                       grid, block,
                                       MAX_THREADS_PER_BLOCK,
                                       MAX_BLOCKS_PER_GRID_DIM);
    }

    if ( numberOfThreads <= static_cast<double>(MAX_THREADS_PER_BLOCK * (MAX_BLOCKS_PER_GRID_DIM * MAX_BLOCKS_PER_GRID_DIM))) {

        return  mwGetLaunchParameters2D(numberOfThreads,
                                        grid, block,
                                        MAX_THREADS_PER_BLOCK,
                                        MAX_BLOCKS_PER_GRID_DIM);
    }

    return mwGetLaunchParameters3D(numberOfThreads,
                                   grid,
                                   block,
                                   MAX_THREADS_PER_BLOCK,
                                   MAX_BLOCKS_PER_GRID_DIM);
    
}

bool mwGetLaunchParameters1D(double numberOfThreads,
                             dim3 *grid, dim3 *block,
                             MW_LAUNCH_UNSIGNED_TYPE MAX_THREADS_PER_BLOCK,
                             MW_LAUNCH_UNSIGNED_TYPE MAX_BLOCKS_PER_GRID_DIM) {

    MW_LAUNCH_UNSIGNED_TYPE x ;
    mwResetDim3(grid);
    mwResetDim3(block);
    
    if ( numberOfThreads <= (double)(512*MAX_BLOCKS_PER_GRID_DIM)) {

        if ( numberOfThreads <= (double) 512)
            x = (MW_LAUNCH_UNSIGNED_TYPE) numberOfThreads;
        else
            x = 512ul;
    }
    else
    {
        x = MAX_THREADS_PER_BLOCK ;
    }
    
#if 0
    unsigned long int x = (numberOfThreads <=  (double)(512*MAX_BLOCKS_PER_GRID_DIM))  ?
        ( (unsigned long int)numberOfThreads <= 512ul ? (unsigned long int)numberOfThreads : 512ul) : MAX_THREADS_PER_BLOCK ;  

#endif
    
    x = mwRoundToMultipleOf32(x);
    block->x  = x;
    grid->x  = (unsigned long int)  ceil(((double)numberOfThreads) / ((double) x));
                     
    return mwValidDim3(block) && mwValidDim3(grid);
    
}


bool mwGetLaunchParameters2D(double numberOfThreads, dim3 *grid, dim3 *block, MW_LAUNCH_UNSIGNED_TYPE MAX_THREADS_PER_BLOCK, MW_LAUNCH_UNSIGNED_TYPE MAX_BLOCKS_PER_GRID_DIM) {

    mwResetDim3(grid);
    mwResetDim3(block);

    block->x = 32;
    block->y = 32;
    block->z = 1;
      
    unsigned long int numBlocksPerDim = (unsigned long int)(
        ceil(sqrt(static_cast<double>(numberOfThreads)/1024)));
    grid->x = numBlocksPerDim;
    grid->y = numBlocksPerDim;
    grid->z =  1;
    
    return mwValidDim3(block) && mwValidDim3(grid);
}

bool mwGetLaunchParameters3D(double numberOfThreads, dim3 *grid, dim3 *block, MW_LAUNCH_UNSIGNED_TYPE MAX_THREADS_PER_BLOCK, MW_LAUNCH_UNSIGNED_TYPE MAX_BLOCKS_PER_GRID_DIM) {

    mwResetDim3(grid);
    mwResetDim3(block);

    numberOfThreads = numberOfThreads < (double) MAX_THREADS_PER_BLOCK * (double) MAX_BLOCKS_PER_GRID_DIM ? numberOfThreads : (double) MAX_THREADS_PER_BLOCK * (double) MAX_BLOCKS_PER_GRID_DIM; 

    block->x = 32;
    block->y = 32;
    block->z = 1;


    unsigned long int numBlocksPerDim = (unsigned long int)(
        ceil(cbrt(static_cast<double>(numberOfThreads)/1024)));
    grid->x = numBlocksPerDim;
    grid->y = numBlocksPerDim;
    grid->z =  numBlocksPerDim;

    return mwValidDim3(block) && mwValidDim3(block);

}

bool mwValidDim3(dim3 *obj) {

    return obj->x > 0  && obj->y > 0 && obj->z > 0 ;
}

void mwResetDim3(dim3 *obj) {
    obj->x = 1;
    obj->y = 1;
    obj->z = 1;
}


void mwResetDim3ToZeros(dim3 *obj) {
    obj->x = 0;
    obj->y = 0;
    obj->z = 0;
}


MW_LAUNCH_UNSIGNED_TYPE mwRoundToMultipleOf32(MW_LAUNCH_UNSIGNED_TYPE val) {
    return ((val + 31ul) / 32ul) * 32ul;
}
