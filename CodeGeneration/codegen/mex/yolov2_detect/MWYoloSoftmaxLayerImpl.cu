#include "MWYoloSoftmaxLayer.hpp"
#include "MWYoloSoftmaxLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
 MWYoloSoftmaxLayerImpl::MWYoloSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int IbSWJNMuIiKbocfQKqXb, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl) , 
eybNKlJCSDUvsznWynwK(IbSWJNMuIiKbocfQKqXb) { 
createYoloSoftmaxLayer(outbufIdx); } 
MWYoloSoftmaxLayerImpl::~MWYoloSoftmaxLayerImpl() { } void 
MWYoloSoftmaxLayerImpl::createYoloSoftmaxLayer(int outbufIdx) { MWTensor* 
ipTensor = getLayer()->getInputTensor(0); MWTensor* opTensor = 
getLayer()->getOutputTensor(0); if (outbufIdx < 0) { float* buffer = NULL; 
CUDA_CALL(hipMalloc((void**)&buffer, sizeof(float) * opTensor->getHeight() * 
opTensor->getWidth() * opTensor->getChannels() * opTensor->getBatchSize())); 
opTensor->setData(buffer); } else { 
opTensor->setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(0))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(),  
opTensor->getChannels(),  opTensor->getHeight(),  opTensor->getWidth()));  int 
fOpFYwKNwIfWjnPzNuob = ipTensor->getChannels() / eybNKlJCSDUvsznWynwK; 
tGsvtyAVkrDznETdweDC = new hipdnnTensorDescriptor_t; 
CUDNN_CALL(hipdnnCreateTensorDescriptor(tGsvtyAVkrDznETdweDC)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*tGsvtyAVkrDznETdweDC, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
fOpFYwKNwIfWjnPzNuob, eybNKlJCSDUvsznWynwK, opTensor->getWidth() * 
opTensor->getHeight())); } void MWYoloSoftmaxLayerImpl::predict() { MWTensor* 
ipTensor = getLayer()->getInputTensor(0); MWTensor* opTensor = 
getLayer()->getOutputTensor(0); CUDNN_CALL(hipdnnSoftmaxForward( 
*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), HIPDNN_SOFTMAX_ACCURATE, 
HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), *tGsvtyAVkrDznETdweDC, 
ipTensor->getData(), getZeroPtr(), *tGsvtyAVkrDznETdweDC, 
opTensor->getData())); } void MWYoloSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*tGsvtyAVkrDznETdweDC)); 
for (int idx = 0; idx < getLayer()->getNumOutputs(); idx++) { float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { if 
(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) { CUDA_FREE_CALL(data); 
} } } }