#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include <stdexcept>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t SUleyRyvAggTFnSdxLru; void 
curand_call_line_file(hiprandStatus_t puSFZkRJmyuFPfQRswDK, const int 
bDTIjtxZiSHtjwzgEluE, const char *RQSttSyDKXCHDWSijmNk) { if (puSFZkRJmyuFPfQRswDK != 
HIPRAND_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"%d at line: %d, file: %s\n", puSFZkRJmyuFPfQRswDK, bDTIjtxZiSHtjwzgEluE, 
RQSttSyDKXCHDWSijmNk); throw std::runtime_error(buffer); } }
#endif
 float* malloc_call_line_file(size_t msize, const int bDTIjtxZiSHtjwzgEluE, const 
char *RQSttSyDKXCHDWSijmNk) { float * mem = (float*)malloc(msize); if (!mem) { char 
buffer[100]; int numElem = sprintf(buffer, "%s at line: %d, file: %s\n", 
"Memory allocation failed. ", bDTIjtxZiSHtjwzgEluE, RQSttSyDKXCHDWSijmNk); throw 
std::runtime_error(buffer); } return mem; } void call_cuda_free(float* mem, 
const int bDTIjtxZiSHtjwzgEluE, const char *RQSttSyDKXCHDWSijmNk) { if (!mem) { return; } 
hipError_t puSFZkRJmyuFPfQRswDK = hipFree(mem); if ((puSFZkRJmyuFPfQRswDK != 
hipErrorDeinitialized) && (puSFZkRJmyuFPfQRswDK != hipSuccess)) { char 
buffer[100]; int numElem = sprintf(buffer, 
"Cuda Error %d(%s) thrown for cuda free call at line: %d, file: %s\n", 
puSFZkRJmyuFPfQRswDK, hipGetErrorString(puSFZkRJmyuFPfQRswDK), bDTIjtxZiSHtjwzgEluE, 
RQSttSyDKXCHDWSijmNk); throw std::runtime_error(buffer); } } void 
cuda_call_line_file(hipError_t puSFZkRJmyuFPfQRswDK, const int bDTIjtxZiSHtjwzgEluE, 
const char *RQSttSyDKXCHDWSijmNk) { if (puSFZkRJmyuFPfQRswDK != hipSuccess) { char 
buffer[100]; int numElem = sprintf(buffer, 
"Cuda Error %d(%s) at line: %d, file: %s\n", puSFZkRJmyuFPfQRswDK, 
hipGetErrorString(puSFZkRJmyuFPfQRswDK), bDTIjtxZiSHtjwzgEluE, RQSttSyDKXCHDWSijmNk); throw 
std::runtime_error(buffer); } } void cudnn_call_line_file(hipdnnStatus_t 
puSFZkRJmyuFPfQRswDK, const int bDTIjtxZiSHtjwzgEluE, const char *RQSttSyDKXCHDWSijmNk) { if 
(puSFZkRJmyuFPfQRswDK != HIPDNN_STATUS_SUCCESS) { char buffer[100]; int numElem = 
sprintf(buffer, "CuDNN Error %d(%s) at line: %d, file: %s\n", 
puSFZkRJmyuFPfQRswDK, hipdnnGetErrorString(puSFZkRJmyuFPfQRswDK), bDTIjtxZiSHtjwzgEluE, 
RQSttSyDKXCHDWSijmNk); throw std::runtime_error(buffer); } } const char* 
cublasGetErrorString(hipblasStatus_t puSFZkRJmyuFPfQRswDK) { 
switch(puSFZkRJmyuFPfQRswDK) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t puSFZkRJmyuFPfQRswDK, const int 
bDTIjtxZiSHtjwzgEluE, const char *RQSttSyDKXCHDWSijmNk) { if (puSFZkRJmyuFPfQRswDK != 
HIPBLAS_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"CuBlas Error %d(%s) at line: %d, file: %s\n", puSFZkRJmyuFPfQRswDK, 
cublasGetErrorString(puSFZkRJmyuFPfQRswDK), bDTIjtxZiSHtjwzgEluE, RQSttSyDKXCHDWSijmNk); 
throw std::runtime_error(buffer); } } 
MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl) : RAtlBpdedvgxUsgDTsch(0.0), QhTesEEIHwhNmHSeYbRR(1.0), 
QhTWatiCfcWYsHdkcyhZ(-1.0), atVCyzqXZAZxwlkRLBRA(layer), 
enPbWLzEmxYCBmzGJutZ(ntwk_impl), PtkeOkuClHzhOfpmBevf(0)  { } 
MWCNNLayerImpl::~MWCNNLayerImpl() { for(std::map<int, 
hipdnnTensorDescriptor_t*>::iterator it = lHtftnmGBvlSSoGOXVui.begin(); it != 
lHtftnmGBvlSSoGOXVui.end(); ++it) { delete it->second; it->second = 0; } } 
float* MWCNNLayerImpl::getZeroPtr() { return &RAtlBpdedvgxUsgDTsch; } float* 
MWCNNLayerImpl::getOnePtr() { return &QhTesEEIHwhNmHSeYbRR; } float* 
MWCNNLayerImpl::getNegOnePtr() { return &QhTWatiCfcWYsHdkcyhZ; } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getOutputDescriptor(int index) { 
std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lHtftnmGBvlSSoGOXVui.find(index); if (it == lHtftnmGBvlSSoGOXVui.end()) { 
hipdnnTensorDescriptor_t* tmp = new hipdnnTensorDescriptor_t;  
lHtftnmGBvlSSoGOXVui[index] = tmp; return tmp; } else { return it->second; } } 
hipdnnTensorDescriptor_t* MWCNNLayerImpl::getOutputDescriptorWithErrorCheck(int 
index) { std::map<int, hipdnnTensorDescriptor_t*>::iterator it = 
lHtftnmGBvlSSoGOXVui.find(index); assert(it != lHtftnmGBvlSSoGOXVui.end()); 
return it->second; } hipdnnTensorDescriptor_t* 
MWCNNLayerImpl::getCuDNNDescriptor(MWTensor* tensor) { MWCNNLayerImpl* impl = 
tensor->getOwner()->getImpl(); if (!impl || 
dynamic_cast<MWPassthroughLayer*>(tensor->getOwner())) { 
assert(dynamic_cast<MWPassthroughLayer*>(tensor->getOwner())); return 
getCuDNNDescriptor(tensor->getOwner()->getInputTensor(0)); } return 
impl->getOutputDescriptorWithErrorCheck(tensor->getSourcePortIndex()); } void 
__global__ __launch_bounds__(1024) padInputImpl(float* in, int inputH, int 
inputW, int inputCh, int outputH, int outputW, int offsetH, int offsetW, float* 
out, int inputElems) { for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
inputElems; i+= blockDim.x*gridDim.x) { int idxB = i/(inputH*inputW*inputCh); 
int rem = (i - idxB*(inputH*inputW*inputCh)); int idxCh = rem/(inputH*inputW); 
int rem1 = rem - idxCh*(inputH*inputW); int idxH = rem1/inputW; int idxCol = 
rem1 - idxH*inputW; if ((idxH < inputH) && (idxCol < inputW)) { int outputR = 
idxH + offsetH; int outputCol = idxCol + offsetW; int outputCh = inputCh; *(out 
+ idxB*(outputH*outputW*outputCh) + idxCh*(outputH*outputW) + outputR*(outputW) 
+ outputCol) = *(in + i); } } } void MWCNNLayerImpl::padInput(float* 
URgvgDXnZskIYGdtimcU, int YGiQICncmsGZkNUyiQyg, int YNDVziqpDddiXQKYZZhX, int 
XVcMnvCXvZpKICKIjgZi, int leWFtIPrKkXLixGWBGJW, int lteHjcLsItGbVPMQtGDB, int 
gzSTokDHvkXefhiGDcWL, int hDaNSVZAofAENeIAiWEw, float* kFQQPKSOkZeHlmrkAXuE, int 
fhikqqlnUKCjleVKDqiG) { int tnTPxeDjBsqLAPkJcPJX = (fhikqqlnUKCjleVKDqiG + 
31)/32 * 32; tnTPxeDjBsqLAPkJcPJX = (tnTPxeDjBsqLAPkJcPJX < 1024) ? 
tnTPxeDjBsqLAPkJcPJX : 1024; int MNuwXDSoGEYeABeVTwOh = 
(fhikqqlnUKCjleVKDqiG + tnTPxeDjBsqLAPkJcPJX - 
1)/tnTPxeDjBsqLAPkJcPJX; padInputImpl<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>(URgvgDXnZskIYGdtimcU, YGiQICncmsGZkNUyiQyg, 
YNDVziqpDddiXQKYZZhX, XVcMnvCXvZpKICKIjgZi, leWFtIPrKkXLixGWBGJW, lteHjcLsItGbVPMQtGDB, 
gzSTokDHvkXefhiGDcWL, hDaNSVZAofAENeIAiWEw, kFQQPKSOkZeHlmrkAXuE, fhikqqlnUKCjleVKDqiG); } 
MWInputLayerImpl::MWInputLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int eVAFqeShtGZAZluKdMvQ, int TaAJDyqFVJXfAfCJhOuU, int vIWQzNvYZSuxmOTVDFhU, int 
NNhshzQGJHLSGjDiVerE, int wMySyzzledUmSLTWhuYH, const char* avg_file_name, int outbufIdx) 
: MWCNNLayerImpl(layer, ntwk_impl) , GFggoMvRWucDMqzlWzCl(0) { 
createInputLayer(eVAFqeShtGZAZluKdMvQ, TaAJDyqFVJXfAfCJhOuU, vIWQzNvYZSuxmOTVDFhU, NNhshzQGJHLSGjDiVerE, 
wMySyzzledUmSLTWhuYH, avg_file_name, outbufIdx); } 
MWInputLayerImpl::~MWInputLayerImpl() { } void 
MWInputLayerImpl::createInputLayer(int eVAFqeShtGZAZluKdMvQ, int TaAJDyqFVJXfAfCJhOuU, int 
vIWQzNvYZSuxmOTVDFhU, int NNhshzQGJHLSGjDiVerE, int wMySyzzledUmSLTWhuYH, const char* 
avg_file_name, int outbufIdx){ if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU*NNhshzQGJHLSGjDiVerE*eVAFqeShtGZAZluKdMvQ)); } else { 
setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
getLayer()->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&KCudOrFMfgCzUPMcdePX)); 
aPzBTLIjCXEQZUlbxayX = wMySyzzledUmSLTWhuYH; 
enPbWLzEmxYCBmzGJutZ->setWorkSpaceSize(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, eVAFqeShtGZAZluKdMvQ, NNhshzQGJHLSGjDiVerE, TaAJDyqFVJXfAfCJhOuU, 
vIWQzNvYZSuxmOTVDFhU)); if( aPzBTLIjCXEQZUlbxayX == 1) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(KCudOrFMfgCzUPMcdePX, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, NNhshzQGJHLSGjDiVerE, TaAJDyqFVJXfAfCJhOuU, vIWQzNvYZSuxmOTVDFhU)); 
CUDA_CALL(hipMalloc((void**)&GFggoMvRWucDMqzlWzCl, 
sizeof(float)*NNhshzQGJHLSGjDiVerE*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU)); int eqOmMKQRpqBqRQCnJmxt = 
NNhshzQGJHLSGjDiVerE*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU;  loadAvg(avg_file_name, 
eqOmMKQRpqBqRQCnJmxt); } else if (aPzBTLIjCXEQZUlbxayX == 2){ 
CUDA_CALL(hipMalloc((void**)&GFggoMvRWucDMqzlWzCl, sizeof(float)*NNhshzQGJHLSGjDiVerE)); int 
eqOmMKQRpqBqRQCnJmxt = NNhshzQGJHLSGjDiVerE;  loadAvg(avg_file_name, eqOmMKQRpqBqRQCnJmxt); }
#ifdef RANDOM
 hiprandGenerateUniform(SUleyRyvAggTFnSdxLru, MW_data, eVAFqeShtGZAZluKdMvQ*NNhshzQGJHLSGjDiVerE*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU);
#endif
 enPbWLzEmxYCBmzGJutZ->setWorkSpaceSize(0); return; } void 
MWInputLayerImpl::loadAvg(const char* RVrPByQXdKmunRZHKWJD, int eqOmMKQRpqBqRQCnJmxt) 
{ FILE* SUjIWYfjMcdzSZaCSVRT = MWCNNLayer::openBinaryFile(RVrPByQXdKmunRZHKWJD); 
assert(SUjIWYfjMcdzSZaCSVRT); float* MdSWZSOAjugbWppryHbR = 
MALLOC_CALL(sizeof(float)*eqOmMKQRpqBqRQCnJmxt); call_fread(MdSWZSOAjugbWppryHbR, 
sizeof(float), eqOmMKQRpqBqRQCnJmxt, SUjIWYfjMcdzSZaCSVRT, RVrPByQXdKmunRZHKWJD); 
CUDA_CALL(hipMemcpy(GFggoMvRWucDMqzlWzCl, MdSWZSOAjugbWppryHbR, 
sizeof(float)*eqOmMKQRpqBqRQCnJmxt, hipMemcpyHostToDevice)); 
free(MdSWZSOAjugbWppryHbR); fclose(SUjIWYfjMcdzSZaCSVRT); return; } void __global__ 
__launch_bounds__(1024) subtractMeanPerChannelImpl(float * 
dMxIKDGTITyhdLqIHBLA, float * QMgBqCuvjnbWHWiVPEwn, const int 
fXhhiexIRPLyKXApPmmy, const int fSbUUBgjKRbNXrHrlOLo, const int 
BUOdotSvmFyUWQKMUdra) {  for(int i = blockDim.x * blockIdx.x + threadIdx.x; i < 
BUOdotSvmFyUWQKMUdra; i+= blockDim.x*gridDim.x) {  int idx = static_cast<int>((i % 
fSbUUBgjKRbNXrHrlOLo) / fXhhiexIRPLyKXApPmmy); 
QMgBqCuvjnbWHWiVPEwn[i] -= dMxIKDGTITyhdLqIHBLA[idx]; } } void 
MWInputLayerImpl::predict() { if (!getData()) { char buffer[100]; int numElem = 
sprintf(buffer, "Uninitialized input data"); throw std::runtime_error(buffer); 
} if ( aPzBTLIjCXEQZUlbxayX == 1) 
CUDNN_CALL(hipdnnAddTensor(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
getNegOnePtr(), KCudOrFMfgCzUPMcdePX, GFggoMvRWucDMqzlWzCl, getOnePtr(), 
*getOutputDescriptor(), getData())); else if( aPzBTLIjCXEQZUlbxayX == 2){ 
MWInputLayer* thisLayer = static_cast<MWInputLayer*>(getLayer()); MWTensor* 
opTensor = thisLayer->getOutputTensor(0); int fSKMHAqIghbYYgyIpNDw = 
opTensor->getHeight()*opTensor->getWidth()* 
opTensor->getChannels()*opTensor->getBatchSize(); int 
fXhhiexIRPLyKXApPmmy = opTensor->getHeight() * opTensor->getWidth(); 
int fSbUUBgjKRbNXrHrlOLo = 
fXhhiexIRPLyKXApPmmy*opTensor->getChannels(); int 
tnTPxeDjBsqLAPkJcPJX = (fSKMHAqIghbYYgyIpNDw < 1024) ? fSKMHAqIghbYYgyIpNDw : 
1024; int MNuwXDSoGEYeABeVTwOh = (fSKMHAqIghbYYgyIpNDw + 
tnTPxeDjBsqLAPkJcPJX - 1)/tnTPxeDjBsqLAPkJcPJX; 
subtractMeanPerChannelImpl<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>( GFggoMvRWucDMqzlWzCl, getData(), 
fXhhiexIRPLyKXApPmmy, fSbUUBgjKRbNXrHrlOLo, fSKMHAqIghbYYgyIpNDw); 
} return; } void MWInputLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < atVCyzqXZAZxwlkRLBRA->getNumOutputs(); idx++) {  float* data = 
atVCyzqXZAZxwlkRLBRA->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} } if ( aPzBTLIjCXEQZUlbxayX == 1) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(KCudOrFMfgCzUPMcdePX)); if (GFggoMvRWucDMqzlWzCl) 
{ CUDA_FREE_CALL(GFggoMvRWucDMqzlWzCl); } } return; } 
MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int inPlace, int outbufIdx)  : MWCNNLayerImpl(layer, ntwk_impl) , 
UdmcwaUkepxfZrpdpcAN(inPlace)  { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&nlIRrOJaFuVaywxOqOyb)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createReLULayer(outbufIdx); } MWReLULayerImpl::~MWReLULayerImpl() { } void 
MWReLULayerImpl::createReLULayer(int outbufIdx) { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); MWTensor* ipTensor = 
reluLayer->getInputTensor(0); MWTensor* opTensor = 
reluLayer->getOutputTensor(0); 
CUDNN_CALL(hipdnnSetActivationDescriptor(nlIRrOJaFuVaywxOqOyb, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));  
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); if 
(outbufIdx < 0) { if(UdmcwaUkepxfZrpdpcAN){ PtkeOkuClHzhOfpmBevf = 
getLayer()->getInputTensor()->getData(); } else{ 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*opTensor->getHeight()* 
opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); } } 
else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
reluLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); }  } void 
MWReLULayerImpl::predict() { MWReLULayer* reluLayer = 
static_cast<MWReLULayer*>(getLayer()); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(reluLayer->getInputTensor()); 
CUDNN_CALL(hipdnnActivationForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
nlIRrOJaFuVaywxOqOyb, getOnePtr(), ipDesc, 
reluLayer->getInputTensor()->getData(), getZeroPtr(), *getOutputDescriptor(), 
PtkeOkuClHzhOfpmBevf)); } void MWReLULayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(nlIRrOJaFuVaywxOqOyb)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); MWTensor* op 
= getLayer()->getOutputTensor(0); float* data = op->getData(); if (data) { 
if((op->getopBufIndex() < 0) && !UdmcwaUkepxfZrpdpcAN) CUDA_FREE_CALL(data); } } 
MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, unsigned FwLnexHgxHRquTKmNpoa,  double AFQBkxwYGKLsACiDKwRM,  
double AHqhysOOIgbDpWZoPUFT,  double BRSPqxNffoBYKqpSVHne, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateLRNDescriptor(&edQOkUJIZbwzEeIcCLzG)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createNormLayer(FwLnexHgxHRquTKmNpoa, AFQBkxwYGKLsACiDKwRM, 
AHqhysOOIgbDpWZoPUFT, BRSPqxNffoBYKqpSVHne, outbufIdx); } 
MWNormLayerImpl::~MWNormLayerImpl() { } void MWNormLayerImpl::createNormLayer( 
unsigned FwLnexHgxHRquTKmNpoa,  double AFQBkxwYGKLsACiDKwRM,  double 
AHqhysOOIgbDpWZoPUFT,  double BRSPqxNffoBYKqpSVHne, int outbufIdx) { MWNormLayer* normLayer 
= static_cast<MWNormLayer*>(getLayer()); MWTensor* ipTensor = 
normLayer->getInputTensor(0); MWTensor* opTensor = 
normLayer->getOutputTensor(0); int numOutputFeatures = opTensor->getChannels(); 
if (outbufIdx < 0) { CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*opTensor->getHeight()*opTensor->getWidth()*numOutputFeatures*opTensor->getBatchSize())); 
} else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
normLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnSetLRNDescriptor(edQOkUJIZbwzEeIcCLzG, 
FwLnexHgxHRquTKmNpoa, AFQBkxwYGKLsACiDKwRM, AHqhysOOIgbDpWZoPUFT, 
BRSPqxNffoBYKqpSVHne)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); return; 
} void MWNormLayerImpl::predict() { MWNormLayer* normLayer = 
static_cast<MWNormLayer*>(getLayer()); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(normLayer->getInputTensor()); 
CUDNN_CALL(hipdnnLRNCrossChannelForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
edQOkUJIZbwzEeIcCLzG, HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), ipDesc, 
normLayer->getInputTensor()->getData(),getZeroPtr(), *getOutputDescriptor(), 
normLayer->getOutputTensor()->getData())); } void MWNormLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyLRNDescriptor(edQOkUJIZbwzEeIcCLzG)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  MWTensor* op = 
getLayer()->getOutputTensor(idx); float* data = op->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0 ) 
CUDA_FREE_CALL(data); } }  } void __global__ MWSetDyForBackPropImpl(float * 
QwUuNuQNtlPXrIwRNiSZ, const int fvTCtkwXgyScJYogJVFU); void __global__ 
doMWMaxPoolingLayerImpl(float * WIxRBCJtmETvfxpuRuus, float * 
VFKMunbyHoAmpHUSkuUn, const int BUOdotSvmFyUWQKMUdra); 
MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DSsxcjIrUgZCKZovyNQf,  int DqxLTLaJwwgQqmrtCDuu,  
int FpguQZSermqZCMRiUfML,  int FrpxvsDMwwgbpqHXWxmN, int CpMjJjtGOeWOzwxpAAQP, int 
ClEhcJFlvGCgiavziIag,  int CufLFODQDXTAPyRqYodN, int DRzwhbNPpftRRIXXfHzd, 
bool GZGFVDrXwFLJleoTDywO, int fylVqSnTjNbHDtlPhzaj, const std::vector<int>& 
MUmglsoWcEiRiAZsclur) : MWCNNLayerImpl(layer, ntwk_impl) , 
BLjrjqvCcCommiXWQLjs(GZGFVDrXwFLJleoTDywO) , WIxRBCJtmETvfxpuRuus(0) 
, QwUuNuQNtlPXrIwRNiSZ(0) , VFKMunbyHoAmpHUSkuUn(0)  {  
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mbKaFvmHqfBiTISNPGKJ)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createMaxPoolingLayer(DSsxcjIrUgZCKZovyNQf,DqxLTLaJwwgQqmrtCDuu,FpguQZSermqZCMRiUfML,FrpxvsDMwwgbpqHXWxmN,CpMjJjtGOeWOzwxpAAQP,ClEhcJFlvGCgiavziIag,CufLFODQDXTAPyRqYodN,DRzwhbNPpftRRIXXfHzd, 
fylVqSnTjNbHDtlPhzaj, MUmglsoWcEiRiAZsclur); } 
MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() { } void 
MWMaxPoolingLayerImpl::createMaxPoolingLayer(int DSsxcjIrUgZCKZovyNQf,  int 
DqxLTLaJwwgQqmrtCDuu,  int FpguQZSermqZCMRiUfML, int FrpxvsDMwwgbpqHXWxmN, int 
CpMjJjtGOeWOzwxpAAQP, int ClEhcJFlvGCgiavziIag,  int CufLFODQDXTAPyRqYodN, 
int DRzwhbNPpftRRIXXfHzd, int fylVqSnTjNbHDtlPhzaj, const std::vector<int>& 
MUmglsoWcEiRiAZsclur) { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
maxpoolLayer->getInputTensor(0); int muwRQxtWMMXAPxSuMYBw = 
CpMjJjtGOeWOzwxpAAQP; int nDsbARncmIrIaLubvLVZ = 
CufLFODQDXTAPyRqYodN; hipdnnTensorDescriptor_t YgcpEBUCwCLaPhyntIio = 
*getCuDNNDescriptor(ipTensor);  
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mbKaFvmHqfBiTISNPGKJ, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, DSsxcjIrUgZCKZovyNQf, DqxLTLaJwwgQqmrtCDuu, 
muwRQxtWMMXAPxSuMYBw, nDsbARncmIrIaLubvLVZ, FpguQZSermqZCMRiUfML, 
FrpxvsDMwwgbpqHXWxmN)); int eVAFqeShtGZAZluKdMvQ, NNhshzQGJHLSGjDiVerE, TaAJDyqFVJXfAfCJhOuU, 
vIWQzNvYZSuxmOTVDFhU; CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(mbKaFvmHqfBiTISNPGKJ, 
YgcpEBUCwCLaPhyntIio, &eVAFqeShtGZAZluKdMvQ ,&NNhshzQGJHLSGjDiVerE, &TaAJDyqFVJXfAfCJhOuU, 
&vIWQzNvYZSuxmOTVDFhU)); TaAJDyqFVJXfAfCJhOuU = getLayer()->getOutputTensor(0)->getHeight(); 
vIWQzNvYZSuxmOTVDFhU = getLayer()->getOutputTensor(0)->getWidth(); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, eVAFqeShtGZAZluKdMvQ, NNhshzQGJHLSGjDiVerE, TaAJDyqFVJXfAfCJhOuU, 
vIWQzNvYZSuxmOTVDFhU)); int outbufIdx = MUmglsoWcEiRiAZsclur[0]; if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*eVAFqeShtGZAZluKdMvQ*NNhshzQGJHLSGjDiVerE*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU)); } else { 
setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
maxpoolLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } if 
(BLjrjqvCcCommiXWQLjs){ 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(1))); const int 
fSKMHAqIghbYYgyIpNDw = 
(ipTensor->getHeight())*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
CUDA_CALL(hipMalloc((void**)&WIxRBCJtmETvfxpuRuus, 
sizeof(float)*fSKMHAqIghbYYgyIpNDw)); assert(fylVqSnTjNbHDtlPhzaj == 2); int 
bufIndex = MUmglsoWcEiRiAZsclur[1]; if (bufIndex < 0) { 
CUDA_CALL(hipMalloc((void**)&VFKMunbyHoAmpHUSkuUn, 
sizeof(float)*eVAFqeShtGZAZluKdMvQ*NNhshzQGJHLSGjDiVerE*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU)); } else { 
VFKMunbyHoAmpHUSkuUn = enPbWLzEmxYCBmzGJutZ->memBuffer[bufIndex]; 
maxpoolLayer->getOutputTensor(1)->setopBufIndex(bufIndex); } 
assert((NNhshzQGJHLSGjDiVerE == ipTensor->getChannels()) && (eVAFqeShtGZAZluKdMvQ == 
ipTensor->getBatchSize()));  const int fvTCtkwXgyScJYogJVFU = 
vIWQzNvYZSuxmOTVDFhU*TaAJDyqFVJXfAfCJhOuU*NNhshzQGJHLSGjDiVerE*eVAFqeShtGZAZluKdMvQ; 
CUDA_CALL(hipMalloc((void**)&QwUuNuQNtlPXrIwRNiSZ, 
sizeof(float)*fvTCtkwXgyScJYogJVFU)); int tnTPxeDjBsqLAPkJcPJX = 
(fvTCtkwXgyScJYogJVFU < 1024) ? fvTCtkwXgyScJYogJVFU : 1024; int 
MNuwXDSoGEYeABeVTwOh = (fvTCtkwXgyScJYogJVFU + tnTPxeDjBsqLAPkJcPJX - 
1)/tnTPxeDjBsqLAPkJcPJX; 
MWSetDyForBackPropImpl<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>( QwUuNuQNtlPXrIwRNiSZ, fvTCtkwXgyScJYogJVFU); } } void 
MWMaxPoolingLayerImpl::predict() { MWMaxPoolingLayer* maxpoolLayer = 
static_cast<MWMaxPoolingLayer*>(getLayer()); hipdnnTensorDescriptor_t 
YgcpEBUCwCLaPhyntIio = *getCuDNNDescriptor(maxpoolLayer->getInputTensor()); 
MWTensor* ipTensor = getLayer()->getInputTensor(0); 
CUDNN_CALL(hipdnnPoolingForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
mbKaFvmHqfBiTISNPGKJ, getOnePtr(), YgcpEBUCwCLaPhyntIio, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(), 
maxpoolLayer->getOutputTensor()->getData())); if (BLjrjqvCcCommiXWQLjs) { 
CUDNN_CALL(hipdnnPoolingBackward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
mbKaFvmHqfBiTISNPGKJ, getOnePtr(), *getOutputDescriptor(0), 
getLayer()->getOutputTensor(0)->getData(), *getOutputDescriptor(0), 
QwUuNuQNtlPXrIwRNiSZ, YgcpEBUCwCLaPhyntIio, ipTensor->getData(), getZeroPtr(), 
YgcpEBUCwCLaPhyntIio, WIxRBCJtmETvfxpuRuus)); int fSKMHAqIghbYYgyIpNDw = 
ipTensor->getHeight()*(ipTensor->getWidth())*(ipTensor->getChannels())*(ipTensor->getBatchSize()); 
int tnTPxeDjBsqLAPkJcPJX = (fSKMHAqIghbYYgyIpNDw < 1024) ? 
fSKMHAqIghbYYgyIpNDw : 1024; int MNuwXDSoGEYeABeVTwOh = (fSKMHAqIghbYYgyIpNDw + 
tnTPxeDjBsqLAPkJcPJX - 1)/tnTPxeDjBsqLAPkJcPJX; 
doMWMaxPoolingLayerImpl<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>( WIxRBCJtmETvfxpuRuus, 
maxpoolLayer->getOutputTensor(1)->getData(), fSKMHAqIghbYYgyIpNDw); } return; } 
void MWMaxPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mbKaFvmHqfBiTISNPGKJ)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(BLjrjqvCcCommiXWQLjs){ 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(1))); } for(int 
idx = 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} } if (WIxRBCJtmETvfxpuRuus){ 
CUDA_FREE_CALL(WIxRBCJtmETvfxpuRuus); } if (QwUuNuQNtlPXrIwRNiSZ){ 
CUDA_FREE_CALL(QwUuNuQNtlPXrIwRNiSZ); }  } float* 
MWMaxPoolingLayerImpl::getIndexData()  { return VFKMunbyHoAmpHUSkuUn; } void 
__global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float * 
QwUuNuQNtlPXrIwRNiSZ, const int fvTCtkwXgyScJYogJVFU) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < fvTCtkwXgyScJYogJVFU; i+= blockDim.x*gridDim.x) { 
QwUuNuQNtlPXrIwRNiSZ[i] = i+1; } } void __global__ __launch_bounds__(1024) 
doMWMaxPoolingLayerImpl(float * WIxRBCJtmETvfxpuRuus, float * 
VFKMunbyHoAmpHUSkuUn, const int BUOdotSvmFyUWQKMUdra) { for(int i = blockDim.x * 
blockIdx.x + threadIdx.x; i < BUOdotSvmFyUWQKMUdra; i+= blockDim.x*gridDim.x) { if 
(static_cast<int>(WIxRBCJtmETvfxpuRuus[i]) != 0){ 
VFKMunbyHoAmpHUSkuUn[static_cast<int>(WIxRBCJtmETvfxpuRuus[i])-1] = 
i; } } } MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int CDJtexcMbXMWAmnNZsNf, const char* 
vpXxoeEhdEosLSsYXkNG,  const char* MIBnYCbKBdUrlfqlHdoo, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MEmIeGILUZNEWEagSzRk)); 
createFCLayer(CDJtexcMbXMWAmnNZsNf, vpXxoeEhdEosLSsYXkNG, 
MIBnYCbKBdUrlfqlHdoo, outbufIdx); } MWFCLayerImpl::~MWFCLayerImpl() { } void 
MWFCLayerImpl::createFCLayer( int CDJtexcMbXMWAmnNZsNf, const char* 
vpXxoeEhdEosLSsYXkNG, const char* MIBnYCbKBdUrlfqlHdoo, int outbufIdx) { 
MWFCLayer* fcLayer = static_cast<MWFCLayer*>(getLayer()); MWTensor* opTensor = 
fcLayer->getOutputTensor(0); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*fcLayer->getOutputTensor()->getBatchSize()*fcLayer->getOutputTensor()->getChannels())); 
} else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
fcLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float)*CDJtexcMbXMWAmnNZsNf* 
fcLayer->getOutputTensor()->getChannels())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 
fcLayer->getOutputTensor()->getBatchSize(),fcLayer->getOutputTensor()->getChannels(), 
1, 1)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(MEmIeGILUZNEWEagSzRk, 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 
fcLayer->getOutputTensor()->getChannels(), 1, 1)); 
CUDA_CALL(hipMalloc((void**)&KHClOltUSuqFVVErSxVb, sizeof(float)*fcLayer->getOutputTensor()->getChannels()));
#ifdef RANDOM
 hiprandGenerateNormal(SUleyRyvAggTFnSdxLru, vIWQzNvYZSuxmOTVDFhU, 
fcLayer->getInputTensor()->getChannels()*fcLayer->getInputTensor()->getWidth()*fcLayer->getInputTensor()->getHeight()*fcLayer->getOutputTensor()->getChannels(), 
0, 0.1); hiprandGenerateNormal(SUleyRyvAggTFnSdxLru, KHClOltUSuqFVVErSxVb, 
fcLayer->getOutputTensor()->getChannels(), -0.5, 1);
#endif
 int eqOmMKQRpqBqRQCnJmxt = CDJtexcMbXMWAmnNZsNf*opTensor->getChannels();  
loadWeights(eqOmMKQRpqBqRQCnJmxt, vpXxoeEhdEosLSsYXkNG); 
loadBias(MIBnYCbKBdUrlfqlHdoo); return; } void MWFCLayerImpl::loadWeights(int 
eqOmMKQRpqBqRQCnJmxt, const char* RVrPByQXdKmunRZHKWJD) {  MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); FILE* SUjIWYfjMcdzSZaCSVRT = 
MWCNNLayer::openBinaryFile(RVrPByQXdKmunRZHKWJD); assert(SUjIWYfjMcdzSZaCSVRT); float* 
MdSWZSOAjugbWppryHbR = MALLOC_CALL(sizeof(float)*eqOmMKQRpqBqRQCnJmxt); 
call_fread(MdSWZSOAjugbWppryHbR, sizeof(float), eqOmMKQRpqBqRQCnJmxt, SUjIWYfjMcdzSZaCSVRT, 
RVrPByQXdKmunRZHKWJD); if( ipTensor->getHeight() != 1 && ipTensor->getWidth() != 
1 ) { float* MgAiRWiTutoTMxKXjmHQ = 
MALLOC_CALL(sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()); for(int 
k=0; k<eqOmMKQRpqBqRQCnJmxt/ipTensor->getHeight()/ipTensor->getWidth(); k++) { 
for(int i=0; i<ipTensor->getHeight()*ipTensor->getWidth(); i++) 
MgAiRWiTutoTMxKXjmHQ[i]=MdSWZSOAjugbWppryHbR[k*ipTensor->getHeight()*ipTensor->getWidth()+i]; 
for(int j=0; j<ipTensor->getHeight(); j++) for(int i=0; i<ipTensor->getWidth(); 
i++) 
MdSWZSOAjugbWppryHbR[k*ipTensor->getHeight()*ipTensor->getWidth()+j*ipTensor->getWidth()+i]=MgAiRWiTutoTMxKXjmHQ[j+i*ipTensor->getHeight()]; 
} free(MgAiRWiTutoTMxKXjmHQ); } CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
MdSWZSOAjugbWppryHbR, sizeof(float)*eqOmMKQRpqBqRQCnJmxt, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", RVrPByQXdKmunRZHKWJD, eqOmMKQRpqBqRQCnJmxt, MdSWZSOAjugbWppryHbR[0]);
#endif
 free(MdSWZSOAjugbWppryHbR); fclose(SUjIWYfjMcdzSZaCSVRT); return; } void 
MWFCLayerImpl::loadBias(const char* RVrPByQXdKmunRZHKWJD) { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* opTensor = 
fcLayer->getOutputTensor(0); FILE* SUjIWYfjMcdzSZaCSVRT = 
MWCNNLayer::openBinaryFile(RVrPByQXdKmunRZHKWJD); assert(SUjIWYfjMcdzSZaCSVRT); int 
eqOmMKQRpqBqRQCnJmxt = opTensor->getChannels();  float* MdSWZSOAjugbWppryHbR = 
MALLOC_CALL(sizeof(float)*eqOmMKQRpqBqRQCnJmxt); call_fread(MdSWZSOAjugbWppryHbR, 
sizeof(float), eqOmMKQRpqBqRQCnJmxt, SUjIWYfjMcdzSZaCSVRT, RVrPByQXdKmunRZHKWJD); 
CUDA_CALL(hipMemcpy(KHClOltUSuqFVVErSxVb, MdSWZSOAjugbWppryHbR, 
sizeof(float)*eqOmMKQRpqBqRQCnJmxt, hipMemcpyHostToDevice)); 
free(MdSWZSOAjugbWppryHbR); fclose(SUjIWYfjMcdzSZaCSVRT); return; } void 
MWFCLayerImpl::predict() { MWFCLayer* fcLayer = 
static_cast<MWFCLayer*>(getLayer()); MWTensor* ipTensor = 
fcLayer->getInputTensor(0); MWTensor* opTensor = fcLayer->getOutputTensor(0); 
int CDJtexcMbXMWAmnNZsNf = 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth(); int 
CGbFsczkgkhjcHoCKzBx = opTensor->getChannels(); int UzaGmBLFEwmwaFXebUma=1; 
int VCbcPxtPsBLTrHYdEvqn=1; if( opTensor->getBatchSize()==1 ) { 
CUDA_CALL(hipMemcpy(getData(), KHClOltUSuqFVVErSxVb, 
sizeof(float)*CGbFsczkgkhjcHoCKzBx, hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*enPbWLzEmxYCBmzGJutZ->getCublasHandle(), HIPBLAS_OP_T, 
CDJtexcMbXMWAmnNZsNf, CGbFsczkgkhjcHoCKzBx, getOnePtr(), 
vIWQzNvYZSuxmOTVDFhU, CDJtexcMbXMWAmnNZsNf, ipTensor->getData(), 
UzaGmBLFEwmwaFXebUma, getOnePtr(),getData(), VCbcPxtPsBLTrHYdEvqn)); } else { 
CUBLAS_CALL(hipblasSgemm(*enPbWLzEmxYCBmzGJutZ->getCublasHandle(), HIPBLAS_OP_T, 
HIPBLAS_OP_N, CGbFsczkgkhjcHoCKzBx, opTensor->getBatchSize(), 
CDJtexcMbXMWAmnNZsNf, getOnePtr(), vIWQzNvYZSuxmOTVDFhU, 
CDJtexcMbXMWAmnNZsNf, ipTensor->getData(), CDJtexcMbXMWAmnNZsNf, 
getZeroPtr(),getData(), CGbFsczkgkhjcHoCKzBx)); 
CUDNN_CALL(hipdnnAddTensor(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), getOnePtr(), 
MEmIeGILUZNEWEagSzRk, KHClOltUSuqFVVErSxVb, getOnePtr(), 
*getOutputDescriptor(),getData())); } return; } void MWFCLayerImpl::cleanup() { 
if (vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); }  
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MEmIeGILUZNEWEagSzRk)); if 
(KHClOltUSuqFVVErSxVb) { CUDA_FREE_CALL(KHClOltUSuqFVVErSxVb); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} } } MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int outbufIdx) : MWCNNLayerImpl(layer, 
ntwk_impl)  {  CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createSoftmaxLayer(outbufIdx); } MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() { } 
void MWSoftmaxLayerImpl::createSoftmaxLayer(int outbufIdx) { MWSoftmaxLayer* 
sfmxLayer = static_cast<MWSoftmaxLayer*>(getLayer()); MWTensor* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensor* opTensor = 
sfmxLayer->getOutputTensor(0); int numOutputFeatures = ipTensor->getChannels(); 
if (outbufIdx < 0) { CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()*numOutputFeatures*ipTensor->getBatchSize())); 
} else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth()));  
return; } void MWSoftmaxLayerImpl::predict() { MWSoftmaxLayer* sfmxLayer = 
static_cast<MWSoftmaxLayer*>(getLayer()); MWTensor* ipTensor = 
sfmxLayer->getInputTensor(0); MWTensor* opTensor = 
sfmxLayer->getOutputTensor(0); hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(ipTensor);  
CUDNN_CALL(hipdnnSoftmaxForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), ipDesc, 
ipTensor->getData(), getZeroPtr(), *getOutputDescriptor(), getData())); } void 
MWSoftmaxLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} } } MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int DSsxcjIrUgZCKZovyNQf,  int DqxLTLaJwwgQqmrtCDuu,  
int FpguQZSermqZCMRiUfML,  int FrpxvsDMwwgbpqHXWxmN,  int CZNYmBcNFSZWvaCklqeM,  int 
CqtPRJvHlGJFssiPzsOm, int outbufIdx) : MWCNNLayerImpl(layer, ntwk_impl)  { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&mbKaFvmHqfBiTISNPGKJ)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createAvgPoolingLayer(DSsxcjIrUgZCKZovyNQf, DqxLTLaJwwgQqmrtCDuu, FpguQZSermqZCMRiUfML, 
FrpxvsDMwwgbpqHXWxmN, CZNYmBcNFSZWvaCklqeM, CqtPRJvHlGJFssiPzsOm, outbufIdx); } 
MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() { } void 
MWAvgPoolingLayerImpl::createAvgPoolingLayer(int DSsxcjIrUgZCKZovyNQf, int 
DqxLTLaJwwgQqmrtCDuu, int FpguQZSermqZCMRiUfML, int FrpxvsDMwwgbpqHXWxmN, int 
CZNYmBcNFSZWvaCklqeM, int CqtPRJvHlGJFssiPzsOm, int outbufIdx) { 
MWAvgPoolingLayer* avgpoolLayer = static_cast<MWAvgPoolingLayer*>(getLayer()); 
MWTensor* ipTensor = avgpoolLayer->getInputTensor(0); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(mbKaFvmHqfBiTISNPGKJ, 
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, HIPDNN_NOT_PROPAGATE_NAN, 
DSsxcjIrUgZCKZovyNQf, DqxLTLaJwwgQqmrtCDuu, CZNYmBcNFSZWvaCklqeM, CqtPRJvHlGJFssiPzsOm, 
FpguQZSermqZCMRiUfML, FrpxvsDMwwgbpqHXWxmN)); int eVAFqeShtGZAZluKdMvQ, NNhshzQGJHLSGjDiVerE, 
TaAJDyqFVJXfAfCJhOuU, vIWQzNvYZSuxmOTVDFhU;  hipdnnTensorDescriptor_t YgcpEBUCwCLaPhyntIio = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(mbKaFvmHqfBiTISNPGKJ, 
YgcpEBUCwCLaPhyntIio, &eVAFqeShtGZAZluKdMvQ ,&NNhshzQGJHLSGjDiVerE, &TaAJDyqFVJXfAfCJhOuU, 
&vIWQzNvYZSuxmOTVDFhU)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, eVAFqeShtGZAZluKdMvQ, NNhshzQGJHLSGjDiVerE, TaAJDyqFVJXfAfCJhOuU, 
vIWQzNvYZSuxmOTVDFhU)); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*eVAFqeShtGZAZluKdMvQ*NNhshzQGJHLSGjDiVerE*TaAJDyqFVJXfAfCJhOuU*vIWQzNvYZSuxmOTVDFhU)); } else { 
setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
avgpoolLayer->getOutputTensor(0)->setopBufIndex(outbufIdx); } } void 
MWAvgPoolingLayerImpl::predict() { MWAvgPoolingLayer* avgpoolLayer = 
static_cast<MWAvgPoolingLayer*>(getLayer()); MWTensor* ipTensor = 
avgpoolLayer->getInputTensor(0); MWTensor* opTensor = 
avgpoolLayer->getOutputTensor(0); hipdnnTensorDescriptor_t YgcpEBUCwCLaPhyntIio = 
*getCuDNNDescriptor(ipTensor); 
CUDNN_CALL(hipdnnPoolingForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
mbKaFvmHqfBiTISNPGKJ, getOnePtr(), YgcpEBUCwCLaPhyntIio, ipTensor->getData(), 
getZeroPtr(), *getOutputDescriptor(),opTensor->getData())); } void 
MWAvgPoolingLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(mbKaFvmHqfBiTISNPGKJ)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); for(int idx = 
0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} } } MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int ) : MWCNNLayerImpl(layer, ntwk_impl) { 
createOutputLayer(); } MWOutputLayerImpl::~MWOutputLayerImpl() { } void 
MWOutputLayerImpl::createOutputLayer() { MWOutputLayer* opLayer = 
static_cast<MWOutputLayer*>(getLayer()); MWTensor* ipTensor = 
opLayer->getInputTensor(0); setData(ipTensor->getData()); return; } void 
MWOutputLayerImpl::predict() { } void MWOutputLayerImpl::cleanup() { }