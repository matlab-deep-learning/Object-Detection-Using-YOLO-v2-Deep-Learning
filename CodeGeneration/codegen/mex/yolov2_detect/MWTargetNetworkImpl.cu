#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWCNNLayerImpl.hpp"
#include <cassert>
 void MWTargetNetworkImpl::preSetup(int BufSize,int numBufsToAlloc) { numBufs = 
numBufsToAlloc; for(int i = 0; i < numBufs; i++) { float *memPtr = 0; 
CUDA_CALL(hipMalloc((void**)&memPtr, sizeof(float)*BufSize)); 
memBuffer.push_back(memPtr); } OumvfgWXDdmsQaciHMHx = new hipblasHandle_t; 
hipblasCreate(OumvfgWXDdmsQaciHMHx); OwenhowBxTAXHXmJpIKd = new 
hipdnnHandle_t; hipdnnCreate(OwenhowBxTAXHXmJpIKd); } void 
MWTargetNetworkImpl::postSetup(MWCNNLayer* layers[],int numLayers) { 
createWorkSpace(&wXLECKaOWaQNZlVHfnNP); for (int i = 0; i < numLayers; i++) { 
layers[i]->postSetup();  } if ((ECTnqgWHyHCHCLBZlffd != (long 
int)*getWorkSpaceSize() && ECTnqgWHyHCHCLBZlffd > -1)) { 
setWorkSpaceSize(ECTnqgWHyHCHCLBZlffd); if (wXLECKaOWaQNZlVHfnNP) 
{ hipFree(wXLECKaOWaQNZlVHfnNP); wXLECKaOWaQNZlVHfnNP = 0; } if 
(*getWorkSpaceSize() > 0){ CUDA_CALL(hipMalloc((void**)&wXLECKaOWaQNZlVHfnNP, 
*getWorkSpaceSize())); } }  if (!wXLECKaOWaQNZlVHfnNP && (*getWorkSpaceSize() > 
0)) { throw std::runtime_error("Out of memory. Unable to allocate workspace."); 
} } void MWTargetNetworkImpl::createWorkSpace(float** wtNPjzxHKNoJIigzXrEl) 
{ hipError_t puSFZkRJmyuFPfQRswDK = hipMalloc((void**)wtNPjzxHKNoJIigzXrEl, 
npGnQZLrEfVTQnEbwqij); if (puSFZkRJmyuFPfQRswDK != hipSuccess) { 
*wtNPjzxHKNoJIigzXrEl = 0; setWorkSpaceSize(0); } } void 
MWTargetNetworkImpl::setWorkSpaceSize(size_t wss) { npGnQZLrEfVTQnEbwqij 
= wss;  } size_t* MWTargetNetworkImpl::getWorkSpaceSize() { return 
&npGnQZLrEfVTQnEbwqij; } float* MWTargetNetworkImpl::getWorkSpace() { 
return wXLECKaOWaQNZlVHfnNP; } long int* 
MWTargetNetworkImpl::getPostSetupWorkSpaceSize() { return 
&ECTnqgWHyHCHCLBZlffd; } void 
MWTargetNetworkImpl::setPostSetupWorkSpaceSize(long int psWSize) { 
ECTnqgWHyHCHCLBZlffd = psWSize; } hipblasHandle_t* 
MWTargetNetworkImpl::getCublasHandle() { return OumvfgWXDdmsQaciHMHx; } 
hipdnnHandle_t* MWTargetNetworkImpl::getCudnnHandle() { return 
OwenhowBxTAXHXmJpIKd; } void MWTargetNetworkImpl::setAutoTune(bool 
autotune) { MW_autoTune = autotune; } bool MWTargetNetworkImpl::getAutoTune() 
const { return MW_autoTune; } void MWTargetNetworkImpl::cleanup() { if 
(wXLECKaOWaQNZlVHfnNP) { hipFree(wXLECKaOWaQNZlVHfnNP); } if 
(OumvfgWXDdmsQaciHMHx) { hipblasDestroy(*OumvfgWXDdmsQaciHMHx); } if 
(OwenhowBxTAXHXmJpIKd) { hipdnnDestroy(*OwenhowBxTAXHXmJpIKd); } for(int 
i = 0; i < memBuffer.size(); i++) { float *memPtr = memBuffer[i]; if(memPtr) { 
CUDA_FREE_CALL(memPtr); }  } }