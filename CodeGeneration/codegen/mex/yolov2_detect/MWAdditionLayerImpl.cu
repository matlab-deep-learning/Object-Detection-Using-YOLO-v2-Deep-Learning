#include "hip/hip_runtime.h"
#include "MWAdditionLayer.hpp"
#include "MWAdditionLayerImpl.hpp"
#include <stdarg.h>
#include <cassert>
 MWAdditionLayerImpl::MWAdditionLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int outbufIdx)  : MWCNNLayerImpl(layer, 
ntwk_impl)  { createAdditionLayer(outbufIdx); } 
MWAdditionLayerImpl::~MWAdditionLayerImpl() { } void 
MWAdditionLayerImpl::createAdditionLayer(int outbufIdx) { MWAdditionLayer* 
AdditionLayer = static_cast<MWAdditionLayer*>(getLayer()); MWTensor* ipTensor = 
AdditionLayer->getInputTensor(0); MWTensor* opTensor = 
AdditionLayer->getOutputTensor(0); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels()*ipTensor->getBatchSize())); 
} else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(0))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(0), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(), 
ipTensor->getChannels(), ipTensor->getHeight(), ipTensor->getWidth())); return 
; } void __global__ __launch_bounds__(1024) addImpl(float* in1, float* in2, 
float* out, size_t maxElems) { size_t i = blockDim.x * blockIdx.x + 
threadIdx.x; for (; i < maxElems; i += size_t(blockDim.x*gridDim.x)) { out[i] = 
in1[i] + in2[i]; } } void MWAdditionLayerImpl::predict() { MWAdditionLayer* 
AdditionLayer = static_cast<MWAdditionLayer*>(getLayer()); MWTensor* ipTensor = 
AdditionLayer->getInputTensor(0);  MWTensor* opTensor = 
AdditionLayer->getOutputTensor(0); int fSKMHAqIghbYYgyIpNDw = 
ipTensor->getHeight()*ipTensor->getWidth()*ipTensor->getChannels()*ipTensor->getBatchSize(); 
hipMemcpy(opTensor->getData(), ipTensor->getData(), 
fSKMHAqIghbYYgyIpNDw*sizeof(float), hipMemcpyDeviceToDevice); for (int k = 1; k 
< AdditionLayer->getNumInputs(); k++) { hipdnnTensorDescriptor_t ipDesc = 
*getCuDNNDescriptor(AdditionLayer->getInputTensor(k)); 
hipdnnAddTensor(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), getOnePtr(), ipDesc, 
AdditionLayer->getInputTensor(k)->getData(), getOnePtr(), 
*getOutputDescriptor(0), AdditionLayer->getOutputTensor(0)->getData() ); } } 
void MWAdditionLayerImpl::cleanup() { MWAdditionLayer* AdditionLayer = 
static_cast<MWAdditionLayer*>(getLayer()); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(0))); for(int idx 
= 0; idx < AdditionLayer->getNumOutputs(); idx++) {  MWTensor* op = 
AdditionLayer->getOutputTensor(idx); float* data = op->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} }  }