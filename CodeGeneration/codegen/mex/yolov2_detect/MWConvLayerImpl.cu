#include "MWConvLayerImpl.hpp"
#include "MWConvLayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include <cassert>
#include <stdio.h>
 MWConvLayerImpl::MWConvLayerImpl(MWCNNLayer* layer, MWTargetNetworkImpl* 
ntwk_impl, int filt_H, int filt_W, int numGrps, int numChnls, int numFilts, int 
FpguQZSermqZCMRiUfML, int FrpxvsDMwwgbpqHXWxmN, int CpMjJjtGOeWOzwxpAAQP, int 
ClEhcJFlvGCgiavziIag, int CufLFODQDXTAPyRqYodN, int DRzwhbNPpftRRIXXfHzd, 
int ATEikvMQPqBefhJzjzhc, int AuqaQHxmPQSyYRemQvyX, const char* 
vpXxoeEhdEosLSsYXkNG, const char* MIBnYCbKBdUrlfqlHdoo, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl)  , yPBlKhIGljihkXaXbYpB(NULL) , vIWQzNvYZSuxmOTVDFhU(NULL) , 
KHClOltUSuqFVVErSxVb(NULL) , veFyKKHbdqBIvQLYBqfF(NULL) , URgvgDXnZskIYGdtimcU(NULL) , 
XCLDbxHBtWRStETWIkId(NULL) , aLsOwwcceEmRSYzllBNs(0) , 
AwZQzUhuWVLGrWgLHRuM(filt_H) , AzTsxYcYjIEJsGQbeYHm (filt_W) , 
BuyZFXzwOMxcePIbCLfl (numGrps) , BdqURaHPmdnfzvtUvocl (numChnls) , 
BlRIQPyqJZORKENzSdYf (numFilts) { enPbWLzEmxYCBmzGJutZ = ntwk_impl; 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&ONvcEjLBnVNUdjMKOAwF)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&SDWKEQTZaTFZByPlzUDR)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MEmIeGILUZNEWEagSzRk)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createConvLayer(FpguQZSermqZCMRiUfML, FrpxvsDMwwgbpqHXWxmN, CpMjJjtGOeWOzwxpAAQP, 
ClEhcJFlvGCgiavziIag, CufLFODQDXTAPyRqYodN, DRzwhbNPpftRRIXXfHzd, 
ATEikvMQPqBefhJzjzhc, AuqaQHxmPQSyYRemQvyX, 
vpXxoeEhdEosLSsYXkNG, MIBnYCbKBdUrlfqlHdoo, outbufIdx); } float 
MWConvLayerImpl::getIsGrouped() { return aLsOwwcceEmRSYzllBNs; } void 
MWConvLayerImpl::setIsGrouped(float ig) { aLsOwwcceEmRSYzllBNs = ig; } void 
MWConvLayerImpl::setOutput2(float* out2) { yPBlKhIGljihkXaXbYpB = out2; } float* 
MWConvLayerImpl::getOutput2() { return yPBlKhIGljihkXaXbYpB; } 
hipdnnTensorDescriptor_t* MWConvLayerImpl::getGroupDescriptor() { return 
&THfVbcZJtANcLKxEriuV; } void MWConvLayerImpl::createConvLayer(int 
FpguQZSermqZCMRiUfML, int FrpxvsDMwwgbpqHXWxmN, int CpMjJjtGOeWOzwxpAAQP, int 
ClEhcJFlvGCgiavziIag , int CufLFODQDXTAPyRqYodN, int DRzwhbNPpftRRIXXfHzd, 
int ATEikvMQPqBefhJzjzhc, int AuqaQHxmPQSyYRemQvyX, const char* 
vpXxoeEhdEosLSsYXkNG, const char* MIBnYCbKBdUrlfqlHdoo, int outbufIdx) { 
MWTensor* ipTensor = getLayer()->getInputTensor(0); int 
OVOphSOolqRQDDoKPwxy = CpMjJjtGOeWOzwxpAAQP; int 
OiVqrkNdXioJhALWMMvm = CufLFODQDXTAPyRqYodN; if 
((CpMjJjtGOeWOzwxpAAQP != ClEhcJFlvGCgiavziIag) || (CufLFODQDXTAPyRqYodN != 
DRzwhbNPpftRRIXXfHzd)) { float* newInput; int inputH = ipTensor->getHeight() + 
CpMjJjtGOeWOzwxpAAQP + ClEhcJFlvGCgiavziIag; int inputW = 
ipTensor->getWidth() + CufLFODQDXTAPyRqYodN + DRzwhbNPpftRRIXXfHzd; 
CUDA_CALL(hipMalloc((void**)&newInput, sizeof(float)*ipTensor->getBatchSize() 
* ipTensor->getChannels() * inputH * inputW)); CUDA_CALL(hipMemset(newInput, 
0, 
sizeof(float)*ipTensor->getBatchSize()*ipTensor->getChannels()*inputH*inputW)); 
URgvgDXnZskIYGdtimcU = new MWTensor(inputH, inputW, ipTensor->getChannels(), 
ipTensor->getBatchSize(), newInput,getLayer(), 0); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&YgcpEBUCwCLaPhyntIio)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(YgcpEBUCwCLaPhyntIio, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, URgvgDXnZskIYGdtimcU->getBatchSize(), URgvgDXnZskIYGdtimcU->getChannels(), 
URgvgDXnZskIYGdtimcU->getHeight(), URgvgDXnZskIYGdtimcU->getWidth())); 
OVOphSOolqRQDDoKPwxy = 0;  OiVqrkNdXioJhALWMMvm = 0;  } else { 
URgvgDXnZskIYGdtimcU = ipTensor; YgcpEBUCwCLaPhyntIio = 
*getCuDNNDescriptor(URgvgDXnZskIYGdtimcU);  } bUVPfnrJhLfHzOLUUrKk = 
CpMjJjtGOeWOzwxpAAQP; cQBKlCKXxecGPJrXBXdk = CufLFODQDXTAPyRqYodN;  
assert(URgvgDXnZskIYGdtimcU != NULL); MWConvLayer* convLayer = static_cast<MWConvLayer*>(getLayer());
#if (CUDNN_MAJOR <= 5)
 { if ((ATEikvMQPqBefhJzjzhc != 1) && (AuqaQHxmPQSyYRemQvyX != 1)){ 
printf("Dilated Convolution only supported for cuDNN 6 or greater "); throw 
std::runtime_error("Unsupported Dilation Factor"); } 
CUDNN_CALL(hipdnnSetConvolution2dDescriptor(ONvcEjLBnVNUdjMKOAwF, 
OVOphSOolqRQDDoKPwxy, OiVqrkNdXioJhALWMMvm, FpguQZSermqZCMRiUfML, 
FrpxvsDMwwgbpqHXWxmN, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(ONvcEjLBnVNUdjMKOAwF, 
OVOphSOolqRQDDoKPwxy, OiVqrkNdXioJhALWMMvm, FpguQZSermqZCMRiUfML, 
FrpxvsDMwwgbpqHXWxmN, ATEikvMQPqBefhJzjzhc, AuqaQHxmPQSyYRemQvyX, 
HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT)); }
#endif
#if (FP16_ENABLED == 1 && ( CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2) ))
 CUDNN_CALL(hipdnnSetConvolutionMathType(ONvcEjLBnVNUdjMKOAwF, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
 int shEncNmxJsMuJKwbrwok, rxMAtVYGgGtZoKBkJcjc; int numInputFeatures = 
BdqURaHPmdnfzvtUvocl*BuyZFXzwOMxcePIbCLfl; int 
fxxCPKTclxXPxrdMAkwi,sRECVoNNtDdcBOWgDyar,sxuOMwKXOKfuExclRaSe; MWTensor* 
opTensor = convLayer->getOutputTensor(0); fxxCPKTclxXPxrdMAkwi = 
opTensor->getChannels(); sRECVoNNtDdcBOWgDyar = opTensor->getHeight(); 
sxuOMwKXOKfuExclRaSe = opTensor->getWidth();  size_t twppmWSuyDzoZjSbrMHi = 0; if( 
BuyZFXzwOMxcePIbCLfl == 1 ) { 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(SDWKEQTZaTFZByPlzUDR, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, fxxCPKTclxXPxrdMAkwi, numInputFeatures, 
AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm));  
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MEmIeGILUZNEWEagSzRk, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, fxxCPKTclxXPxrdMAkwi, 1, 1)); 
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(ONvcEjLBnVNUdjMKOAwF, 
YgcpEBUCwCLaPhyntIio, SDWKEQTZaTFZByPlzUDR, &shEncNmxJsMuJKwbrwok, 
&rxMAtVYGgGtZoKBkJcjc, &sRECVoNNtDdcBOWgDyar, &sxuOMwKXOKfuExclRaSe)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, shEncNmxJsMuJKwbrwok, rxMAtVYGgGtZoKBkJcjc, 
opTensor->getHeight(), opTensor->getWidth())); assert(opTensor->getHeight() == 
sRECVoNNtDdcBOWgDyar); assert(opTensor->getWidth() == sxuOMwKXOKfuExclRaSe);
#if (CUDNN_MAJOR < 7)
 { 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
YgcpEBUCwCLaPhyntIio, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, 
*getOutputDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, 
&NtWaRGCHLeTapjWdEHHS)); }
#else
 { hipdnnConvolutionFwdAlgoPerf_t perf_results[3]; int returnedAlgoCount; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
YgcpEBUCwCLaPhyntIio, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, 
*getOutputDescriptor(), 3, &returnedAlgoCount, perf_results)); 
NtWaRGCHLeTapjWdEHHS = perf_results[0].algo; }
#endif
 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
YgcpEBUCwCLaPhyntIio, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, 
*getOutputDescriptor(), NtWaRGCHLeTapjWdEHHS, &twppmWSuyDzoZjSbrMHi)); } else { 
setIsGrouped(1); MWTensor* ipTensor = URgvgDXnZskIYGdtimcU;  XCLDbxHBtWRStETWIkId = 
ipTensor->getData() + ipTensor->getChannels()/BuyZFXzwOMxcePIbCLfl * 
ipTensor->getHeight() * ipTensor->getWidth(); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&ZCArwzdUdwQuFQUWjnUE)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(ZCArwzdUdwQuFQUWjnUE, 
HIPDNN_DATA_FLOAT, ipTensor->getBatchSize(), 
ipTensor->getChannels()/BuyZFXzwOMxcePIbCLfl, ipTensor->getHeight(), 
ipTensor->getWidth(), 
ipTensor->getChannels()*ipTensor->getHeight()*ipTensor->getWidth(), 
ipTensor->getHeight()*ipTensor->getWidth(), ipTensor->getWidth(), 1)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getGroupDescriptor()));  
CUDNN_CALL(hipdnnSetFilter4dDescriptor(SDWKEQTZaTFZByPlzUDR, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, BlRIQPyqJZORKENzSdYf, BdqURaHPmdnfzvtUvocl, 
AwZQzUhuWVLGrWgLHRuM, AzTsxYcYjIEJsGQbeYHm));  
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(ONvcEjLBnVNUdjMKOAwF, 
ZCArwzdUdwQuFQUWjnUE, SDWKEQTZaTFZByPlzUDR, &shEncNmxJsMuJKwbrwok, 
&rxMAtVYGgGtZoKBkJcjc, &sRECVoNNtDdcBOWgDyar, &sxuOMwKXOKfuExclRaSe)); 
assert(opTensor->getHeight() == sRECVoNNtDdcBOWgDyar); assert(opTensor->getWidth() 
== sxuOMwKXOKfuExclRaSe); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(*getGroupDescriptor(), 
HIPDNN_DATA_FLOAT, shEncNmxJsMuJKwbrwok, rxMAtVYGgGtZoKBkJcjc, sRECVoNNtDdcBOWgDyar, 
sxuOMwKXOKfuExclRaSe, 
rxMAtVYGgGtZoKBkJcjc*BuyZFXzwOMxcePIbCLfl*sRECVoNNtDdcBOWgDyar*sxuOMwKXOKfuExclRaSe, 
sRECVoNNtDdcBOWgDyar*sxuOMwKXOKfuExclRaSe, sxuOMwKXOKfuExclRaSe, 1)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, shEncNmxJsMuJKwbrwok, 
rxMAtVYGgGtZoKBkJcjc*BuyZFXzwOMxcePIbCLfl, sRECVoNNtDdcBOWgDyar, sxuOMwKXOKfuExclRaSe)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MEmIeGILUZNEWEagSzRk, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, rxMAtVYGgGtZoKBkJcjc*BuyZFXzwOMxcePIbCLfl, 1, 1));
#if (CUDNN_MAJOR < 7) 
 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
ZCArwzdUdwQuFQUWjnUE, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, 
*getGroupDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NtWaRGCHLeTapjWdEHHS));
#else
 hipdnnConvolutionFwdAlgoPerf_t perf_results[3]; int returnedAlgoCount; 
CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
ZCArwzdUdwQuFQUWjnUE, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, 
*getGroupDescriptor(), 3, &returnedAlgoCount,perf_results)); 
NtWaRGCHLeTapjWdEHHS = perf_results[0].algo;
#endif
 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
ZCArwzdUdwQuFQUWjnUE, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, 
*getGroupDescriptor(), NtWaRGCHLeTapjWdEHHS, &twppmWSuyDzoZjSbrMHi)); } if( 
twppmWSuyDzoZjSbrMHi > *enPbWLzEmxYCBmzGJutZ->getWorkSpaceSize() ) { 
enPbWLzEmxYCBmzGJutZ->setWorkSpaceSize(twppmWSuyDzoZjSbrMHi); }  
assert(shEncNmxJsMuJKwbrwok == ipTensor->getBatchSize()); 
assert(fxxCPKTclxXPxrdMAkwi == rxMAtVYGgGtZoKBkJcjc * 
BuyZFXzwOMxcePIbCLfl); if (outbufIdx < 0) { 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, sizeof(float) * 
opTensor->getBatchSize() * opTensor->getChannels() * opTensor->getHeight() * 
opTensor->getWidth())); } else { 
setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
getLayer()->getOutputTensor(0)->setopBufIndex(outbufIdx); } 
CUDA_CALL(hipMalloc((void**)&vIWQzNvYZSuxmOTVDFhU, 
sizeof(float)*BdqURaHPmdnfzvtUvocl*fxxCPKTclxXPxrdMAkwi*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm)); 
CUDA_CALL(hipMalloc((void**)&KHClOltUSuqFVVErSxVb, sizeof(float)*fxxCPKTclxXPxrdMAkwi));
#ifdef RANDOM
 hiprandGenerateNormal(SUleyRyvAggTFnSdxLru, vIWQzNvYZSuxmOTVDFhU, 
BdqURaHPmdnfzvtUvocl*fxxCPKTclxXPxrdMAkwi*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm, 
0, 0.1); hiprandGenerateNormal(SUleyRyvAggTFnSdxLru, KHClOltUSuqFVVErSxVb, 
fxxCPKTclxXPxrdMAkwi, -0.5, 1);
#endif
 if( BuyZFXzwOMxcePIbCLfl == 2 ) { veFyKKHbdqBIvQLYBqfF = vIWQzNvYZSuxmOTVDFhU + 
BlRIQPyqJZORKENzSdYf * BdqURaHPmdnfzvtUvocl * AwZQzUhuWVLGrWgLHRuM * 
AzTsxYcYjIEJsGQbeYHm; setOutput2(getData() + fxxCPKTclxXPxrdMAkwi/ 2 
* sRECVoNNtDdcBOWgDyar * sxuOMwKXOKfuExclRaSe); setIsGrouped(1); } 
loadWeights(vpXxoeEhdEosLSsYXkNG); loadBias(MIBnYCbKBdUrlfqlHdoo); } void 
MWConvLayerImpl::predict() { MWConvLayer* convLayer = 
static_cast<MWConvLayer*>(getLayer()); if (URgvgDXnZskIYGdtimcU != 
convLayer->getInputTensor()) { CUDA_CALL(hipMemset(URgvgDXnZskIYGdtimcU->getData(), 
0, 
sizeof(float)*URgvgDXnZskIYGdtimcU->getBatchSize()*URgvgDXnZskIYGdtimcU->getChannels()*URgvgDXnZskIYGdtimcU->getHeight()*URgvgDXnZskIYGdtimcU->getWidth())); 
 int fhikqqlnUKCjleVKDqiG = 
convLayer->getInputTensor()->getHeight()*convLayer->getInputTensor()->getWidth()*convLayer->getInputTensor()->getBatchSize()*convLayer->getInputTensor()->getChannels(); 
MWCNNLayerImpl::padInput(convLayer->getInputTensor()->getData(), 
convLayer->getInputTensor()->getHeight(), 
convLayer->getInputTensor()->getWidth(), 
convLayer->getInputTensor()->getChannels(), URgvgDXnZskIYGdtimcU->getHeight(), 
URgvgDXnZskIYGdtimcU->getWidth(), bUVPfnrJhLfHzOLUUrKk, cQBKlCKXxecGPJrXBXdk, 
URgvgDXnZskIYGdtimcU->getData(), fhikqqlnUKCjleVKDqiG); } if(BuyZFXzwOMxcePIbCLfl == 1 
) { assert(getData() != URgvgDXnZskIYGdtimcU->getData()); 
CUDNN_CALL(hipdnnConvolutionForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(),getOnePtr(), 
YgcpEBUCwCLaPhyntIio, URgvgDXnZskIYGdtimcU->getData(), SDWKEQTZaTFZByPlzUDR, 
vIWQzNvYZSuxmOTVDFhU, ONvcEjLBnVNUdjMKOAwF, NtWaRGCHLeTapjWdEHHS, 
enPbWLzEmxYCBmzGJutZ->getWorkSpace(), *enPbWLzEmxYCBmzGJutZ->getWorkSpaceSize(), 
getZeroPtr(), *getOutputDescriptor(),getData())); 
CUDNN_CALL(hipdnnAddTensor(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), getOnePtr(), 
MEmIeGILUZNEWEagSzRk, KHClOltUSuqFVVErSxVb, getOnePtr(), 
*getOutputDescriptor(),getData())); } else { assert(getData() != 
URgvgDXnZskIYGdtimcU->getData()); 
CUDNN_CALL(hipdnnConvolutionForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
getOnePtr(), ZCArwzdUdwQuFQUWjnUE, URgvgDXnZskIYGdtimcU->getData(), 
SDWKEQTZaTFZByPlzUDR, vIWQzNvYZSuxmOTVDFhU, ONvcEjLBnVNUdjMKOAwF, NtWaRGCHLeTapjWdEHHS, 
enPbWLzEmxYCBmzGJutZ->getWorkSpace(), *enPbWLzEmxYCBmzGJutZ->getWorkSpaceSize(), 
getZeroPtr(), *getGroupDescriptor(), getData())); 
CUDNN_CALL(hipdnnConvolutionForward(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
getOnePtr(), ZCArwzdUdwQuFQUWjnUE, XCLDbxHBtWRStETWIkId, SDWKEQTZaTFZByPlzUDR, 
veFyKKHbdqBIvQLYBqfF, ONvcEjLBnVNUdjMKOAwF, NtWaRGCHLeTapjWdEHHS, 
enPbWLzEmxYCBmzGJutZ->getWorkSpace(), *enPbWLzEmxYCBmzGJutZ->getWorkSpaceSize(), 
getZeroPtr(), *getGroupDescriptor(), getOutput2())); 
CUDNN_CALL(hipdnnAddTensor(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), getOnePtr(), 
MEmIeGILUZNEWEagSzRk, KHClOltUSuqFVVErSxVb, getOnePtr(), *getOutputDescriptor(), 
getData())); } } void MWConvLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(ONvcEjLBnVNUdjMKOAwF)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(SDWKEQTZaTFZByPlzUDR)); if 
(vIWQzNvYZSuxmOTVDFhU) { CUDA_FREE_CALL(vIWQzNvYZSuxmOTVDFhU); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MEmIeGILUZNEWEagSzRk)); if 
(KHClOltUSuqFVVErSxVb) { CUDA_FREE_CALL(KHClOltUSuqFVVErSxVb); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(URgvgDXnZskIYGdtimcU != getLayer()->getInputTensor(0)) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(YgcpEBUCwCLaPhyntIio)); 
CUDA_FREE_CALL(URgvgDXnZskIYGdtimcU->getData()); } if (getIsGrouped()) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(ZCArwzdUdwQuFQUWjnUE));  
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getGroupDescriptor())); } for(int idx 
= 0; idx < getLayer()->getNumOutputs(); idx++) {  float* data = 
getLayer()->getOutputTensor(idx)->getData(); if (data) { 
if(getLayer()->getOutputTensor(idx)->getopBufIndex() < 0) CUDA_FREE_CALL(data); 
} } } void MWConvLayerImpl::loadWeights(const char* RVrPByQXdKmunRZHKWJD) { 
MWConvLayer* convLayer = static_cast<MWConvLayer*>(getLayer()); FILE* 
SUjIWYfjMcdzSZaCSVRT = MWCNNLayer::openBinaryFile(RVrPByQXdKmunRZHKWJD); 
assert(SUjIWYfjMcdzSZaCSVRT); assert(BdqURaHPmdnfzvtUvocl == 
URgvgDXnZskIYGdtimcU->getChannels()/BuyZFXzwOMxcePIbCLfl); int eqOmMKQRpqBqRQCnJmxt = 
BdqURaHPmdnfzvtUvocl*convLayer->getOutputTensor()->getChannels()*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm; 
 float* MdSWZSOAjugbWppryHbR = MALLOC_CALL(sizeof(float)*eqOmMKQRpqBqRQCnJmxt); 
call_fread(MdSWZSOAjugbWppryHbR, sizeof(float), eqOmMKQRpqBqRQCnJmxt, SUjIWYfjMcdzSZaCSVRT, 
RVrPByQXdKmunRZHKWJD); if( AwZQzUhuWVLGrWgLHRuM != 1 && 
AzTsxYcYjIEJsGQbeYHm != 1 ) { float* MgAiRWiTutoTMxKXjmHQ = 
MALLOC_CALL(sizeof(float)*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm); 
for(int k=0; k<eqOmMKQRpqBqRQCnJmxt/AwZQzUhuWVLGrWgLHRuM/AzTsxYcYjIEJsGQbeYHm; 
k++) { for(int i=0; i<AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm; i++) 
MgAiRWiTutoTMxKXjmHQ[i]=MdSWZSOAjugbWppryHbR[k*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm+i]; 
for(int j=0; j<AwZQzUhuWVLGrWgLHRuM; j++) for(int i=0; 
i<AzTsxYcYjIEJsGQbeYHm; i++) 
MdSWZSOAjugbWppryHbR[k*AwZQzUhuWVLGrWgLHRuM*AzTsxYcYjIEJsGQbeYHm+j*AzTsxYcYjIEJsGQbeYHm+i]=MgAiRWiTutoTMxKXjmHQ[j+i*AwZQzUhuWVLGrWgLHRuM]; 
} free(MgAiRWiTutoTMxKXjmHQ); } CUDA_CALL(hipMemcpy(vIWQzNvYZSuxmOTVDFhU, 
MdSWZSOAjugbWppryHbR, sizeof(float)*eqOmMKQRpqBqRQCnJmxt, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", RVrPByQXdKmunRZHKWJD, eqOmMKQRpqBqRQCnJmxt, MdSWZSOAjugbWppryHbR[0]);
#endif
 free(MdSWZSOAjugbWppryHbR); fclose(SUjIWYfjMcdzSZaCSVRT); } void 
MWConvLayerImpl::loadBias(const char* RVrPByQXdKmunRZHKWJD) { MWConvLayer* 
convLayer = static_cast<MWConvLayer*>(getLayer()); FILE* SUjIWYfjMcdzSZaCSVRT = 
MWCNNLayer::openBinaryFile(RVrPByQXdKmunRZHKWJD);  assert(SUjIWYfjMcdzSZaCSVRT); int 
eqOmMKQRpqBqRQCnJmxt = convLayer->getOutputTensor()->getChannels();  float* 
MdSWZSOAjugbWppryHbR = MALLOC_CALL(sizeof(float)*eqOmMKQRpqBqRQCnJmxt); 
call_fread(MdSWZSOAjugbWppryHbR, sizeof(float), eqOmMKQRpqBqRQCnJmxt, SUjIWYfjMcdzSZaCSVRT, 
RVrPByQXdKmunRZHKWJD); CUDA_CALL(hipMemcpy(KHClOltUSuqFVVErSxVb, MdSWZSOAjugbWppryHbR, 
sizeof(float)*eqOmMKQRpqBqRQCnJmxt, hipMemcpyHostToDevice)); 
free(MdSWZSOAjugbWppryHbR); fclose(SUjIWYfjMcdzSZaCSVRT); } void 
MWConvLayerImpl::postSetup() { if(enPbWLzEmxYCBmzGJutZ->getAutoTune()) { 
getConvAlgoTuned(); } else if(!enPbWLzEmxYCBmzGJutZ->getWorkSpace()) { 
getConvAlgoNoWorkSpace();  } hipdnnTensorDescriptor_t tmpInDesc = getIsGrouped() 
? ZCArwzdUdwQuFQUWjnUE : YgcpEBUCwCLaPhyntIio; hipdnnTensorDescriptor_t 
hnewnpwgzKmOdualajhn = getIsGrouped() ? *getGroupDescriptor() : 
*getOutputDescriptor();  size_t twppmWSuyDzoZjSbrMHi; 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
tmpInDesc, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, hnewnpwgzKmOdualajhn, 
NtWaRGCHLeTapjWdEHHS, &twppmWSuyDzoZjSbrMHi)); if( (long int)twppmWSuyDzoZjSbrMHi 
> *enPbWLzEmxYCBmzGJutZ->getPostSetupWorkSpaceSize() ) { 
enPbWLzEmxYCBmzGJutZ->setPostSetupWorkSpaceSize((long int)twppmWSuyDzoZjSbrMHi); 
} } void MWConvLayerImpl::getConvAlgoTuned() { hipdnnConvolutionFwdAlgoPerf_t 
perf_results[3]; hipdnnTensorDescriptor_t tempInDesc = getIsGrouped() ? 
ZCArwzdUdwQuFQUWjnUE : YgcpEBUCwCLaPhyntIio; hipdnnTensorDescriptor_t 
hnewnpwgzKmOdualajhn = getIsGrouped() ? *getGroupDescriptor() : 
*getOutputDescriptor(); int returnedAlgoCount; 
CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
tempInDesc, URgvgDXnZskIYGdtimcU->getData(), SDWKEQTZaTFZByPlzUDR, vIWQzNvYZSuxmOTVDFhU, 
ONvcEjLBnVNUdjMKOAwF, hnewnpwgzKmOdualajhn, getData(), 3, &returnedAlgoCount, 
&perf_results[0], enPbWLzEmxYCBmzGJutZ->getWorkSpace(), 
*enPbWLzEmxYCBmzGJutZ->getWorkSpaceSize())); NtWaRGCHLeTapjWdEHHS = 
perf_results[0].algo; } void MWConvLayerImpl::getConvAlgoNoWorkSpace() { 
assert(enPbWLzEmxYCBmzGJutZ->getWorkSpace() == 0); hipdnnTensorDescriptor_t 
tempInDesc = getIsGrouped() ? ZCArwzdUdwQuFQUWjnUE : YgcpEBUCwCLaPhyntIio; 
hipdnnTensorDescriptor_t hnewnpwgzKmOdualajhn = getIsGrouped() ? 
*getGroupDescriptor() : *getOutputDescriptor(); 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), 
tempInDesc, SDWKEQTZaTFZByPlzUDR, ONvcEjLBnVNUdjMKOAwF, hnewnpwgzKmOdualajhn, 
HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &NtWaRGCHLeTapjWdEHHS));  }