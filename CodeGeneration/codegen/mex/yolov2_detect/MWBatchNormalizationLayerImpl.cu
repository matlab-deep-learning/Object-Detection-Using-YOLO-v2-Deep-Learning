#include "MWBatchNormalizationLayer.hpp"
#include "MWBatchNormalizationLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWCNNLayerImpl.hpp"
#include <stdio.h>
#include <cassert> 
 MWBatchNormalizationLayerImpl::MWBatchNormalizationLayerImpl(MWCNNLayer* 
layer, MWTargetNetworkImpl* ntwk_impl, double const GsZlHFuhbvjLtRMDjXnW, 
const char* IwKnaBoXVubIRYcxEJLH, const char* JgLfgHrHMEMmMYTettJF, const 
char* JsZenQeBPMhwsyEhVHiD, const char* 
JwxFdqOKggeawILBfGgg, int inPlace, int outbufIdx) : 
MWCNNLayerImpl(layer, ntwk_impl)  , oJUVMnJggjhEdQLWzIUC(NULL) , 
gsJtSpgIkTNvahoTFqow(NULL) , ugnnrhsgTeWucrMPCJUc(NULL) , 
ujSEtllBwMdSJhSkFCia(NULL) , UdmcwaUkepxfZrpdpcAN(inPlace) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MRnAxrRZGjgErnCjJcbo)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
createBatchNormalizationLayer(GsZlHFuhbvjLtRMDjXnW, IwKnaBoXVubIRYcxEJLH, 
JgLfgHrHMEMmMYTettJF, JsZenQeBPMhwsyEhVHiD, 
JwxFdqOKggeawILBfGgg, outbufIdx); } 
MWBatchNormalizationLayerImpl::~MWBatchNormalizationLayerImpl() { } void 
MWBatchNormalizationLayerImpl::createBatchNormalizationLayer(double const 
GsZlHFuhbvjLtRMDjXnW, const char* IwKnaBoXVubIRYcxEJLH, const char* 
JgLfgHrHMEMmMYTettJF, const char* JsZenQeBPMhwsyEhVHiD, const char* 
JwxFdqOKggeawILBfGgg, int outbufIdx) { MWBatchNormalizationLayer* 
BNLayer = static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* 
ipTensor = BNLayer->getInputTensor(); MWTensor* opTensor = 
BNLayer->getOutputTensor(); REXdEoRjxuQJkqgIDihy = GsZlHFuhbvjLtRMDjXnW; 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); const 
size_t numBytes = sizeof(float)*ipTensor->getChannels(); 
CUDA_CALL(hipMalloc((void**)&oJUVMnJggjhEdQLWzIUC, numBytes)); 
CUDA_CALL(hipMalloc((void**)&gsJtSpgIkTNvahoTFqow, numBytes)); 
CUDA_CALL(hipMalloc((void**)&ugnnrhsgTeWucrMPCJUc, numBytes)); 
CUDA_CALL(hipMalloc((void**)&ujSEtllBwMdSJhSkFCia, numBytes)); 
eFaDPmxDdzHlRYSAoMmX = HIPDNN_BATCHNORM_SPATIAL; 
CUDNN_CALL(hipdnnDeriveBNTensorDescriptor(MRnAxrRZGjgErnCjJcbo, 
*getOutputDescriptor(), eFaDPmxDdzHlRYSAoMmX));  loadScale(JgLfgHrHMEMmMYTettJF); 
loadOffset(IwKnaBoXVubIRYcxEJLH); 
loadTrainedMean(JsZenQeBPMhwsyEhVHiD); 
loadTrainedVariance(JwxFdqOKggeawILBfGgg); if (outbufIdx < 0) { 
if(UdmcwaUkepxfZrpdpcAN){ PtkeOkuClHzhOfpmBevf = 
getLayer()->getInputTensor()->getData(); } else{ 
CUDA_CALL(hipMalloc((void**)&PtkeOkuClHzhOfpmBevf, 
sizeof(float)*opTensor->getHeight()* 
opTensor->getWidth()*opTensor->getChannels()*opTensor->getBatchSize())); } } 
else { setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } } void 
MWBatchNormalizationLayerImpl::iLoadParamOntoGPU(char const * const 
RqCYCrGsNvzKYrRMXbsI, int const fYaOQTeunPwVjnhhTECh, float* 
SugesRlPIbOVzRgNWRnl) { FILE* SUjIWYfjMcdzSZaCSVRT = 
MWCNNLayer::openBinaryFile(RqCYCrGsNvzKYrRMXbsI); assert(SUjIWYfjMcdzSZaCSVRT); int 
const NMMfJylfQjiIUAKhXCJb = sizeof(float)*fYaOQTeunPwVjnhhTECh; float* 
MdSWZSOAjugbWppryHbR = (float*)malloc(NMMfJylfQjiIUAKhXCJb); 
fread(MdSWZSOAjugbWppryHbR, sizeof(float), fYaOQTeunPwVjnhhTECh, SUjIWYfjMcdzSZaCSVRT); 
fclose(SUjIWYfjMcdzSZaCSVRT); CUDA_CALL(hipMemcpy(SugesRlPIbOVzRgNWRnl, 
MdSWZSOAjugbWppryHbR, NMMfJylfQjiIUAKhXCJb, hipMemcpyHostToDevice)); 
free(MdSWZSOAjugbWppryHbR); } void MWBatchNormalizationLayerImpl::loadScale(const 
char* RqCYCrGsNvzKYrRMXbsI) { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(RqCYCrGsNvzKYrRMXbsI, 
opTensor->getChannels(), oJUVMnJggjhEdQLWzIUC); } void 
MWBatchNormalizationLayerImpl::loadOffset(const char* RqCYCrGsNvzKYrRMXbsI) { 
MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(RqCYCrGsNvzKYrRMXbsI, 
opTensor->getChannels(), gsJtSpgIkTNvahoTFqow); } void 
MWBatchNormalizationLayerImpl::loadTrainedMean(const char* RqCYCrGsNvzKYrRMXbsI) 
{ MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(RqCYCrGsNvzKYrRMXbsI, 
opTensor->getChannels(), ugnnrhsgTeWucrMPCJUc); } void 
MWBatchNormalizationLayerImpl::loadTrainedVariance(const char* 
RqCYCrGsNvzKYrRMXbsI) { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* opTensor = 
BNLayer->getOutputTensor(); iLoadParamOntoGPU(RqCYCrGsNvzKYrRMXbsI, 
opTensor->getChannels(), ujSEtllBwMdSJhSkFCia); } void 
MWBatchNormalizationLayerImpl::predict() { MWBatchNormalizationLayer* BNLayer = 
static_cast<MWBatchNormalizationLayer*>(getLayer()); MWTensor* ipTensor = 
BNLayer->getInputTensor(); MWTensor* opTensor = BNLayer->getOutputTensor(); 
const hipdnnTensorDescriptor_t UWAGLbDcvybdWBtshhsr = 
*getCuDNNDescriptor(ipTensor); float* UpnEytIWGokwbTFkBcSx = ipTensor->getData(); 
hipdnnTensorDescriptor_t kNsviQGMPdXzNMRixGWR = *getOutputDescriptor(); float* 
kkqTyvjYvRFtTOyQUwrF = getData(); 
CUDNN_CALL(hipdnnBatchNormalizationForwardInference( 
*enPbWLzEmxYCBmzGJutZ->getCudnnHandle(), eFaDPmxDdzHlRYSAoMmX, getOnePtr(), 
getZeroPtr(),  UWAGLbDcvybdWBtshhsr, UpnEytIWGokwbTFkBcSx, kNsviQGMPdXzNMRixGWR, 
kkqTyvjYvRFtTOyQUwrF, MRnAxrRZGjgErnCjJcbo, oJUVMnJggjhEdQLWzIUC, 
gsJtSpgIkTNvahoTFqow, ugnnrhsgTeWucrMPCJUc, ujSEtllBwMdSJhSkFCia, 
REXdEoRjxuQJkqgIDihy)); } void MWBatchNormalizationLayerImpl::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MRnAxrRZGjgErnCjJcbo)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); 
if(oJUVMnJggjhEdQLWzIUC) { CUDA_FREE_CALL(oJUVMnJggjhEdQLWzIUC); } 
if(gsJtSpgIkTNvahoTFqow) { CUDA_FREE_CALL(gsJtSpgIkTNvahoTFqow); } 
if(ugnnrhsgTeWucrMPCJUc) { CUDA_FREE_CALL(ugnnrhsgTeWucrMPCJUc); } 
if(ujSEtllBwMdSJhSkFCia) { CUDA_FREE_CALL(ujSEtllBwMdSJhSkFCia); } 
MWTensor* op = getLayer()->getOutputTensor(0); float* data = op->getData(); if 
(data) { if((op->getopBufIndex() < 0) && !UdmcwaUkepxfZrpdpcAN) 
CUDA_FREE_CALL(data); } }