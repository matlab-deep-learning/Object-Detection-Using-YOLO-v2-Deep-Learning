#include "MWYoloExtractionLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <cassert>
 MWYoloExtractionLayerImpl::MWYoloExtractionLayerImpl(MWCNNLayer* layer, 
MWTargetNetworkImpl* ntwk_impl, int IbSWJNMuIiKbocfQKqXb, const 
std::vector<int>& MUmglsoWcEiRiAZsclur) : MWCNNLayerImpl(layer, ntwk_impl) , 
eybNKlJCSDUvsznWynwK(IbSWJNMuIiKbocfQKqXb) , fylVqSnTjNbHDtlPhzaj(3) { 
createYoloExtractionLayer(MUmglsoWcEiRiAZsclur); } 
MWYoloExtractionLayerImpl::~MWYoloExtractionLayerImpl() { } void 
MWYoloExtractionLayerImpl::createYoloExtractionLayer( const std::vector<int>& 
MUmglsoWcEiRiAZsclur) { assert(fylVqSnTjNbHDtlPhzaj == 3); for (int i = 0; i 
< fylVqSnTjNbHDtlPhzaj; i++) { MWTensor* opTensor = 
getLayer()->getOutputTensor(i); int outbufIdx = MUmglsoWcEiRiAZsclur[i]; if 
(outbufIdx < 0) { float* buffer = NULL; CUDA_CALL(hipMalloc((void**)&buffer, 
sizeof(float) * opTensor->getBatchSize() * opTensor->getChannels() * 
opTensor->getHeight() * opTensor->getWidth())); opTensor->setData(buffer); } 
else { opTensor->setData(enPbWLzEmxYCBmzGJutZ->memBuffer[outbufIdx]); 
opTensor->setopBufIndex(outbufIdx); } 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor(i))); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor( *getOutputDescriptor(i), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, opTensor->getBatchSize(), 
opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth())); } } 
void MWYoloExtractionLayerImpl::predict() { assert(this->getData() != 
getLayer()->getInputTensor(0)->getData()); if 
(getLayer()->getInputTensor(0)->getBatchSize() == 1) { float* 
XNZmftADYzuZnIYIpBaT = getLayer()->getInputTensor(0)->getData(); long int 
jscBrjkVJyVfMMDjFpgl = (getLayer()->getOutputTensor(0)->getHeight()) 
* (getLayer()->getOutputTensor(0)->getWidth()); long int 
jhFUWlztBndwjbXwYNaJ = 
getLayer()->getOutputTensor(0)->getChannels() * jscBrjkVJyVfMMDjFpgl; 
CUDA_CALL(hipMemcpy(getLayer()->getOutputTensor(0)->getData(), 
XNZmftADYzuZnIYIpBaT, sizeof(float) * jhFUWlztBndwjbXwYNaJ, 
hipMemcpyDeviceToDevice)); long int jmcFOAbZArjGDNhshSro = 
getLayer()->getOutputTensor(1)->getChannels() * jscBrjkVJyVfMMDjFpgl; 
CUDA_CALL(hipMemcpy(getLayer()->getOutputTensor(1)->getData(), 
XNZmftADYzuZnIYIpBaT + jhFUWlztBndwjbXwYNaJ, sizeof(float) * 
jmcFOAbZArjGDNhshSro, hipMemcpyDeviceToDevice)); long int 
jaqKGCwoANNDMHgAsehk = 
getLayer()->getOutputTensor(2)->getChannels() * jscBrjkVJyVfMMDjFpgl; 
CUDA_CALL(hipMemcpy(getLayer()->getOutputTensor(2)->getData(), 
XNZmftADYzuZnIYIpBaT + jhFUWlztBndwjbXwYNaJ + 
jmcFOAbZArjGDNhshSro, sizeof(float) * 
jaqKGCwoANNDMHgAsehk, hipMemcpyDeviceToDevice)); } else { int 
aFDPITUhkPdupMfPOBnd = getLayer()->getInputTensor(0)->getWidth(); long int 
ZinudJuZuGitiNTsJpBR = (getLayer()->getInputTensor(0)->getHeight()) * 
(getLayer()->getInputTensor(0)->getWidth()); long int 
ZUTPCvgISoRdtnhGqXzM = ZinudJuZuGitiNTsJpBR * 
(getLayer()->getInputTensor(0)->getChannels()); long int ZKjSVYDDjACizBkGbqBq = 
ZUTPCvgISoRdtnhGqXzM * getLayer()->getInputTensor(0)->getBatchSize(); 
long int jhFUWlztBndwjbXwYNaJ = ZinudJuZuGitiNTsJpBR * 
(getLayer()->getOutputTensor(0)->getChannels()); long int 
jmcFOAbZArjGDNhshSro = ZinudJuZuGitiNTsJpBR * 
(getLayer()->getOutputTensor(1)->getChannels()); long int 
jaqKGCwoANNDMHgAsehk = ZinudJuZuGitiNTsJpBR * 
(getLayer()->getOutputTensor(2)->getChannels()); long int rSmEWccbJFfPGddhPemm = 
((ZKjSVYDDjACizBkGbqBq + 31) / 32) * 32; long int tnTPxeDjBsqLAPkJcPJX = 
(rSmEWccbJFfPGddhPemm < 1024) ? rSmEWccbJFfPGddhPemm : 1024; long int 
MNuwXDSoGEYeABeVTwOh = (ZKjSVYDDjACizBkGbqBq + tnTPxeDjBsqLAPkJcPJX - 
1) / tnTPxeDjBsqLAPkJcPJX; YoloExtractionImpl<<<MNuwXDSoGEYeABeVTwOh, 
tnTPxeDjBsqLAPkJcPJX>>>( getLayer()->getInputTensor(0)->getData(), 
getLayer()->getOutputTensor(0)->getData(), 
getLayer()->getOutputTensor(1)->getData(), 
getLayer()->getOutputTensor(2)->getData(), eybNKlJCSDUvsznWynwK, 
aFDPITUhkPdupMfPOBnd, ZinudJuZuGitiNTsJpBR, 
ZUTPCvgISoRdtnhGqXzM, jhFUWlztBndwjbXwYNaJ, 
jmcFOAbZArjGDNhshSro, jaqKGCwoANNDMHgAsehk, 
ZKjSVYDDjACizBkGbqBq); } return; } void MWYoloExtractionLayerImpl::cleanup() { 
for (int idx = 0; idx < getLayer()->getNumOutputs(); idx++) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor(idx))); float* 
data = getLayer()->getOutputTensor(idx)->getData(); if (data) { if 
(getLayer()->getOutputTensor(idx)->getopBufIndex() == -1) { 
CUDA_FREE_CALL(data); } } } }